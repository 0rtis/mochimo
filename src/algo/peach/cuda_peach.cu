#include "hip/hip_runtime.h"
/*
 * cuda_trigg.cu  Multi-GPU CUDA Mining
 *
 * Copyright (c) 2019 by Adequate Systems, LLC.  All Rights Reserved.
 * See LICENSE.PDF   **** NO WARRANTY ****
 *
 * Date: 10 August 2018
 * Revision: 31
 */

#include <stdio.h>
#include <stdint.h>
#include <inttypes.h>
#include <unistd.h>
#include <hip/hip_runtime.h>

#include "../../config.h"
#include "peach.h"
#include "nighthash.cu"

__constant__ static uint8_t __align__(8) c_phash[32];
__constant__ static uint8_t __align__(8) c_input[108];
__constant__ static uint8_t __align__(8) c_difficulty;
__constant__ static int Z_MASS[4] = {238,239,240,242};
__constant__ static int Z_ING[2]  = {42,43};
__constant__ static int Z_TIME[16] =
   {82,83,84,85,86,87,88,243,249,250,251,252,253,254,255,253};
__constant__ static int Z_AMB[16] =
   {77,94,95,96,126,214,217,218,220,222,223,224,225,226,227,228};
__constant__ static int Z_ADJ[64] =
   {61,62,63,64,65,66,67,68,69,70,71,72,73,74,75,76,77,78,79,80,81,82,83,84,85,86,87,
    88,89,90,91,92,94,95,96,97,98,99,100,101,102,103,104,105,107,108,109,110,112,114,
    115,116,117,118,119,120,121,122,123,124,125,126,127,128};

inline int cudaCheckError( const char *msg, uint32_t gpu, const char *file)
{
   hipError_t err = hipGetLastError();
   if(hipSuccess != err) {
      fprintf(stderr, "%s Error (#%d) in %s: %s\n",
              msg, gpu, file, hipGetErrorString(err));
      return 1;
   }
   return 0;
}


__device__ uint32_t cuda_next_index(uint32_t index, uint8_t *g_map, uint8_t *nonce)
{
   CUDA_NIGHTHASH_CTX nighthash;
   byte seed[HASHLEN + 4 + TILE_LENGTH];
   byte hash[HASHLEN];
   int i, seedlen;

   /* Create nighthash seed for this index on the map */
   seedlen = HASHLEN + 4 + TILE_LENGTH;
   memcpy(seed, nonce, HASHLEN);
   memcpy(seed + HASHLEN, (byte *) &index, 4);
   memcpy(seed + HASHLEN + 4, &g_map[index * TILE_LENGTH], TILE_LENGTH);
   
   /* Setup nighthash the seed, NO TRANSFORM */
   cuda_nighthash_init(&nighthash, seed, seedlen, index, 0);

   /* Update nighthash with the seed data */
   cuda_nighthash_update(&nighthash, seed, seedlen);

   /* Finalize nighthash into the first 32 byte chunk of the tile */
   cuda_nighthash_final(&nighthash, hash);

   /* Convert 32-byte Hash Value Into 8x 32-bit Unsigned Integer */
   for(i = 0, index = 0; i < 8; i++)
      index += ((uint32_t *) hash)[i];

   return index & MAP;
}


__device__ void cuda_gen_tile(uint32_t index, uint8_t *g_map)
{
   CUDA_NIGHTHASH_CTX nighthash;
   byte seed[4 + HASHLEN];
   byte *tilep;
   int i, j, seedlen;

   /* Set map pointer */
   tilep = &g_map[index * TILE_LENGTH];

   /* Create nighthash seed for this index on the map */
   seedlen = 4 + HASHLEN;
   memcpy(seed, (byte *) &index, 4);
   memcpy(seed + 4, c_phash, HASHLEN);

   /* Setup nighthash with a transform of the seed */
   cuda_nighthash_init(&nighthash, seed, seedlen, index, 1);

   /* Update nighthash with the seed data */
   cuda_nighthash_update(&nighthash, seed, seedlen);

   /* Finalize nighthash into the first 32 byte chunk of the tile */
   cuda_nighthash_final(&nighthash, tilep);

   /* Begin constructing the full tile */
   for(i = 0; i < TILE_LENGTH; i += HASHLEN) { /* For each tile row */
      /* Set next row's pointer location */
      j = i + HASHLEN;

      /* Hash the current row to the next, if not at the end */
      if(j < TILE_LENGTH) {
         /* Setup nighthash with a transform of the current row */
         cuda_nighthash_init(&nighthash, &tilep[i], HASHLEN, index, 1);

         /* Update nighthash with the seed data and tile index */
         cuda_nighthash_update(&nighthash, &tilep[i], HASHLEN);
         cuda_nighthash_update(&nighthash, (byte *) &index, 4);

         /* Finalize nighthash into the first 32 byte chunk of the tile */
         cuda_nighthash_final(&nighthash, &tilep[j]);
      }
   }
}


__global__ void cuda_build_map(uint8_t *g_map)
{
   const uint32_t thread = blockDim.x * blockIdx.x + threadIdx.x;
   if (thread < MAP)
      cuda_gen_tile(thread, g_map);
}


__global__ void cuda_find_peach(uint32_t threads, uint8_t *g_map,
                                int32_t *g_found, uint8_t *g_seed)
{
   const uint32_t thread = blockDim.x * blockIdx.x + threadIdx.x;

   CUDA_SHA256_CTX ictx;
   uint8_t seed[16] = {0}, nonce[32] = {0};
   uint8_t bt_hash[32], fhash[32];
   int32_t i, j, n, x;
   uint32_t sm;

   if (thread < threads) {
      /* Determine second seed */
      if(thread < 131072) { /* This frame permutations: 131,072 */
         seed[ 0] = Z_TIME[(thread & 15)];
         seed[ 1] = Z_AMB[(thread >> 4) & 15];
         seed[ 2] = 1;
         seed[ 3] = Z_ADJ[(thread >> 8) & 63];
         seed[ 4] = Z_MASS[(thread >> 14) & 3];
         seed[ 5] = 1;
         seed[ 6] = Z_ING[(thread >> 16) & 1];
      }

      /* store full nonce */
      #pragma unroll
      for (i = 0; i < 16; i++)
         nonce[i] = c_input[i + 92];

      #pragma unroll
      for (i = 0; i < 16; i++)
         nonce[i+16] = seed[i];

      /*********************************************************/
      /* Hash 124 bytes of Block Trailer, including both seeds */

      cuda_sha256_init(&ictx);
      cuda_sha256_update(&ictx, c_input, 108);
      cuda_sha256_update(&ictx, seed, 16);
      cuda_sha256_final(&ictx, bt_hash);

      /****************************************************/
      /* Follow the tile path based on the selected nonce */
      
      sm = bt_hash[0];
      #pragma unroll
      for(i = 1; i < HASHLEN; i++)
         sm *= bt_hash[i];
      sm %= MAP;

      /* make <JUMP> tile jumps to find the final tile */
      #pragma unroll
      for(j = 0; j < JUMP; j++)
        sm = cuda_next_index(sm, g_map, nonce);

      /****************************************************************/
      /* Check the hash of the final tile produces the desired result */

      cuda_sha256_init(&ictx);
      cuda_sha256_update(&ictx, bt_hash, HASHLEN);
      cuda_sha256_update(&ictx, &g_map[sm * TILE_LENGTH], TILE_LENGTH);
      cuda_sha256_final(&ictx, fhash);

      /* Evaluate hash */
      for (x = i = j = n = 0; i < HASHLEN; i++) {
         x = fhash[i];
         if (x != 0) {
            for(j = 7; j > 0; j--) {
               x >>= 1;
               if(x == 0) {
                  n += j;
                  break;
               }
            }
            break;
         }
         n += 8;
      }

      if(n >= c_difficulty && !atomicExch(g_found, 1)) {
         /* PRINCESS FOUND! */
         #pragma unroll
         for (i = 0; i < 16; i++)
            g_seed[i] = seed[i];
      }
      /* Our princess is in another castle ! */
   }
}


extern "C" {

typedef struct __peach_cuda_ctx {
   byte init, curr_seed[16], next_seed[16];
   byte *seed, *d_seed;
   byte *input, *d_map;
   int32_t *d_found;
   hipStream_t stream;
} PeachCudaCTX;

/* Max 63 GPUs Supported */
PeachCudaCTX ctx[63];
dim3 grid(512);
dim3 block(256);
uint32_t threads = 131072;
int32_t nGPU = 0;
int32_t *found;
byte gpuInit = 0;
byte bnum[8] = {0};
byte *diff;
byte *phash;

int init_cuda_peach(byte difficulty, byte *prevhash, byte *blocknumber) {
   int i;
   
   /* Obtain and check system GPU count */
   hipGetDeviceCount(&nGPU);
   if(nGPU<1 || nGPU>63) return nGPU;
   
   /* Allocate pinned host memory */
   hipHostMalloc(&found, 4);
   hipHostMalloc(&diff, 1);
   hipHostMalloc(&phash, 32);
   
   /* Copy immediate block data to pinned memory */
   *found = 0;
   *diff = difficulty;
   memcpy(phash, prevhash, 32);
   
   /* Initialize GPU context init variable incase
    * it holds a random number from memory */
   if(gpuInit == 0) {
      gpuInit = 1;
      for (i = 0; i < nGPU; i++)
         ctx[i].init = 0;
   }
   
   /* Initialize GPU data asynchronously */
   for (i = 0; i < nGPU; i++) {
      hipSetDevice(i);
      
      /* Create Stream */
      hipStreamCreate(&ctx[i].stream);
      
      /* Allocate device memory */
      hipMalloc(&ctx[i].d_found, 4);
      hipMalloc(&ctx[i].d_seed, 16);
      
      /* Allocate associated device-host memory */
      hipHostMalloc(&ctx[i].seed, 16);
      hipHostMalloc(&ctx[i].input, 108);
      
      /* Copy immediate block data to device memory */
      hipMemcpyToSymbolAsync(HIP_SYMBOL(c_difficulty), diff, 1, 0,
                              hipMemcpyHostToDevice, ctx[i].stream);
      hipMemcpyToSymbolAsync(HIP_SYMBOL(c_phash), phash, 32, 0,
                              hipMemcpyHostToDevice, ctx[i].stream);
      
      /* Set remaining device memory */
      hipMemsetAsync(ctx[i].d_found, 0, 4, ctx[i].stream);
      hipMemsetAsync(ctx[i].d_seed, 0, 16, ctx[i].stream);
      
      /* Set initial round variables */
      ctx[i].next_seed[0] = 0;
      
      /* If first init, setup map and cache */
      if(ctx[i].init == 0) {
         /* NOTE: The device MAP that holds the data of a map DOES NOT
          * explicitly get free()'d. The reason behind this is because
          * we reuse the map variable between blocks, and just rebuild
          * the map once every block. The GPU free's the MAP when the
          * program ends by default. This can be adjusted later. */
         hipMalloc(&ctx[i].d_map, MAP_LENGTH);
         ctx[i].init = 1;
      }
      
      /* (re)Build map if new block */
      if(memcmp(bnum, blocknumber, 8) != 0)
         cuda_build_map<<<4096, 256, 0, ctx[i].stream>>>(ctx[i].d_map);
   }
   
   /* Check for any GPU initialization errors */
   for(i = 0; i < nGPU; i++) {
      hipSetDevice(i);
      hipStreamSynchronize(ctx[i].stream);
      if(cudaCheckError("init_cuda_peach()", i, __FILE__))
         return -1;
   }
   
   /* Update block number */
   memcpy(bnum, blocknumber, 8);

   return nGPU;
}

void free_cuda_peach() {
   int i;
   
   /* Free pinned host memory */
   hipHostFree(diff);
   hipHostFree(found);
   hipHostFree(phash);
   
   /* Free GPU data */
   for (i = 0; i<nGPU; i++) {
      hipSetDevice(i);
      
      /* Destroy Stream */
      hipStreamDestroy(ctx[i].stream);
      
      /* Free device memory */
      hipFree(ctx[i].d_found);
      hipFree(ctx[i].d_seed);
      
      /* Free associated device-host memory */
      hipHostFree(ctx[i].seed);
      hipHostFree(ctx[i].input);
   }
}

extern byte *trigg_gen(byte *in);

__host__ void cuda_peach(byte *bt, uint32_t *hps, byte *runflag)
{
   int i;
   uint64_t lastnHaiku, nHaiku = 0;
   time_t seconds = time(NULL);
   for( ; *runflag && *found == 0; ) {
      for (i=0; i<nGPU; i++) {
         /* Prepare next seed for GPU... */
         if(ctx[i].next_seed[0] == 0) {
            /* ... generate first GPU seed (and expand as Haiku) */
            trigg_gen(ctx[i].next_seed);

            /* ... and prepare round data */
            memcpy(ctx[i].input, bt, 92);
            memcpy(ctx[i].input+92, ctx[i].next_seed, 16);
         }
         /* Check if GPU has finished */
         hipSetDevice(i);
         if(hipStreamQuery(ctx[i].stream) == hipSuccess) {
            hipMemcpy(found, ctx[i].d_found, 4, hipMemcpyDeviceToHost);
            if(*found==1) { /* SOLVED A BLOCK! */
               hipMemcpy(ctx[i].seed, ctx[i].d_seed, 16, hipMemcpyDeviceToHost);
               memcpy(bt + 92, ctx[i].curr_seed, 16);
               memcpy(bt + 92 + 16, ctx[i].seed, 16);
               break;
            }
            /* Send new GPU round Data */
            hipMemcpyToSymbolAsync(HIP_SYMBOL(c_input), ctx[i].input, 108, 0,
                                    hipMemcpyHostToDevice, ctx[i].stream);
            /* Start GPU round */
            cuda_find_peach<<<grid, block, 0, ctx[i].stream>>>(threads,
                                 ctx[i].d_map, ctx[i].d_found, ctx[i].d_seed);

            /* Add to haiku count */
            nHaiku += threads;

            /* Store round vars aside for checks next loop */
            memcpy(ctx[i].curr_seed,ctx[i].next_seed,16);
            ctx[i].next_seed[0] = 0;
         }
         
         /* Waiting on GPU? ... */
         if(cudaCheckError("cuda_peach()", i, __FILE__)) {
            *runflag = 0;
            return;
         }
      }
      
      /* Chill a bit if nothing is happening */
      if(lastnHaiku == nHaiku) usleep(1000);
      else lastnHaiku = nHaiku;
   }
    
   seconds = time(NULL) - seconds;
   if(seconds == 0) seconds = 1;
   nHaiku /= seconds;
   *hps = (uint32_t) nHaiku;
}


}
