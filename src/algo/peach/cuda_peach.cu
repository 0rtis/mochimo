#include "hip/hip_runtime.h"
/*
 * cuda_trigg.cu  Multi-GPU CUDA Mining
 *
 * Copyright (c) 2018 by Adequate Systems, LLC.  All Rights Reserved.
 * See LICENSE.PDF   **** NO WARRANTY ****
 *
 * Date: 10 August 2018
 * Revision: 31
 *
 * Attribution:  The portions of this code on lines 20 through 233 are work
 * made for hire by a Mochimo Discord user, and are not subject to to Mochimo
 * Cryptocurrency Engine License Agreement.  The remainder of this file below
 * line 233 is subject to the license as found in LICENSE.PDF
 * 
 * Anon Discord User: Let Stack know if you want attribution, and we'll give
 * you a proper credit here.  As of our last conversation you just wanted to
 * be paid with no attribution, which frankly feels a little weird to everyone.
 *
 */

#include <stdio.h>
#include <stdint.h>
#include <inttypes.h>
#include <unistd.h>
#include <hip/hip_runtime.h>
extern "C" {
#include "../../crypto/sha256.h"
}

#include "../../config.h"
#include "sha256.cuh"
#include "peach.h"

#define AS_UINT2(addr) *((uint2*)(addr))

// Define this to turn on error checking
#define CUDA_ERROR_CHECK

#define CudaSafeCall( err ) __cudaSafeCall( err, __FILE__, __LINE__ )
#define CudaCheckError()    __cudaCheckError( __FILE__, __LINE__ )

inline void __cudaSafeCall( hipError_t err, const char *file, const int line )
{
#ifdef CUDA_ERROR_CHECK
    if ( hipSuccess != err )
    {
        fprintf( stderr, "cudaSafeCall() failed at %s:%i : %s\n",
                 file, line, hipGetErrorString( err ) );
        exit( -1 );
    }
#endif

    return;
}

inline void __cudaCheckError( const char *file, const int line )
{
#ifdef CUDA_ERROR_CHECK
    hipError_t err = hipGetLastError();
    if ( hipSuccess != err )
    {
        fprintf( stderr, "cudaCheckError() failed at %s:%i : %s\n",
                 file, line, hipGetErrorString( err ) );
        exit( -1 );
    }

    // More careful checking. However, this will affect performance.
    // Comment away if needed.
    err = hipDeviceSynchronize();
    if( hipSuccess != err )
    {
        fprintf( stderr, "cudaCheckError() with sync failed at %s:%i : %s\n",
                 file, line, hipGetErrorString( err ) );
        exit( -1 );
    }
#endif

    return;
}



__constant__ static uint8_t __align__(8) c_phash[32];
__constant__ static uint8_t __align__(8) c_input32[108];
__constant__ static uint8_t __align__(8) c_difficulty;
__constant__ static int Z_PREP[4]  = {12,13,14,15};
__constant__ static int Y_PREP[2]  = {16,17};
__constant__ static int Z_ING[16]  = {18,19,20,21,22,23,24,25,26,27,28,29,30,31,32,33};
__constant__ static int Y_ING[8]   = {34,35,36,37,38,39,40,41};
__constant__ static int X_ING[2]   = {42,43};
__constant__ static int Z_INF[16]  = {44,45,46,47,48,50,51,52,53,54,55,56,57,58,59,60};
__constant__ static int Z_ADJ[64]  = {61,62,63,64,65,66,67,68,69,70,71,72,73,74,75,76,77,78,79,80,
                                      81,82,83,84,85,86,87,88,89,90,91,92,94,95,96,97,98,99,100,
                                      101,102,103,104,105,107,108,109,110,112,114,115,116,117,118,
                                      119,120,121,122,123,124,125,126,127,128};
__constant__ static int Z_AMB[16]  = {77,94,95,96,126,214,217,218,220,222,223,224,225,226,227,228};
__constant__ static int Z_TIMED[8] = {84,243,249,250,251,252,253,255};
__constant__ static int Z_NS[64]   = {129,130,131,132,133,134,135,136,137,138,145,149,154,155,156,
                                      157,177,178,179,180,182,183,184,185,186,187,188,189,190,191,
                                      192,193,194,196,197,198,199,200,201,202,203,204,205,206,207,
                                      208,209,210,211,212,213,241,244,245,246,247,248,249,250,251,
                                      252,253,254,255};
__constant__ static int Z_NPL[32]  = {139,140,141,142,143,144,146,147,148,150,151,153,158,159,160,
                                      161,162,163,164,165,166,167,168,169,170,171,172,173,174,175,
                                      176,181};
__constant__ static int Z_MASS[16] = {214,215,216,217,218,219,220,221,222,223,224,225,226,227,228,229};
__constant__ static int Y_MASS[8]  = {230,231,232,233,234,235,236,237};
__constant__ static int X_MASS[4]  = {238,239,240,242};
__constant__ static int Z_INGINF[32] = {18,19,20,21,22,25,26,27,28,29,30,36,37,38,39,40,41,42,44,
                                        46,47,48,49,51,52,53,54,55,56,57,58,59};
__constant__ static int Z_TIME[16] = {82,83,84,85,86,87,88,243,249,250,251,252,253,254,255,253};
__constant__ static int Z_INGADJ[64] = {18,19,20,21,22,23,24,25,26,27,28,29,30,31,32,33,34,35,36,
                                        37,38,39,40,41,42,43,23,24,31,32,33,34,61,62,63,64,65,66,
                                        67,68,69,70,71,72,73,74,75,76,77,78,79,80,81,82,83,84,85,
                                        86,87,88,89,90,91,92};



__device__ uint32_t cuda_next_index(uint32_t tilenum, uint8_t *tilep, uint8_t *nonce) {
  /**
   * Assume tile[1024] pointer and nonce[16] pointer
   * Plus an additional unsigned - 0x20A0 bits */
   
  uint32_t index;
  byte hash[HASHLEN];
  cuda_SHA256_CTX ictx;

  cuda_sha256_init(&ictx);
  cuda_sha256_update(&ictx, nonce, HASHLEN);
  cuda_sha256_update(&ictx, (byte*)&tilenum, sizeof(uint32_t));
  cuda_sha256_update(&ictx, tilep, TILE_LENGTH);

  cuda_sha256_final(&ictx, hash);

  index =  *(uint32_t*)&hash[0]; //read first 4 bytes as unsigned int
  index += *(uint32_t*)&hash[4]; //read next 4 bytes as unsigned int

  return index % MAP;
}

__device__ uint8_t * cuda_gen_tile(uint32_t tilenum, uint8_t *phash,
                                   uint8_t *g_map, uint8_t *g_cache) {
  /**
   * Declarations */
  cuda_SHA256_CTX ictx;
  int i, j, k, t, z, exp;
  uint8_t bits, _104, _72, *tilep;
  uint32_t op;
  float *floatp;

  /* set map pointer */
  tilep = &g_map[tilenum * TILE_LENGTH];
  
  /* begin tile data */
  cuda_sha256_init(&ictx);
  cuda_sha256_update(&ictx, phash, HASHLEN);
  cuda_sha256_update(&ictx, (byte*)&tilenum, sizeof(uint32_t));
  cuda_sha256_final(&ictx, tilep);
  
  /* set operation variables */
  _104 = 104;
  _72 = 72;

    for(i = j = k = 0; i < TILE_LENGTH; i+=HASHLEN) //for each row of the tile
    {
      for(op = 0; j < i+HASHLEN; j+=4)
      {

        /* set float pointer */
        floatp = (float*)&tilep[j];
        
        /**
         * Order of operations dependent on initial 8 bits
         * Operations:
         *   1) right shift by 4 to obtain the exponent value
         *   2) 50% chance of exponent being negative
         *   3) 50% chance of changing sign of float */
        if(tilep[k] & 1) {
          k++;
          exp = tilep[k++] >> 4;
          if(tilep[k++] & 1) exp ^= 0x80000000;
          if(tilep[k++] & 1) *floatp = -(*floatp);
        } else
        if(tilep[k] & 2) {
          k++;
          exp = tilep[k++] >> 4;
          if(tilep[k++] & 1) *floatp = -(*floatp);
          if(tilep[k++] & 1) exp ^= 0x80000000;
        } else {
          k++;
          if(tilep[k++] & 1) *floatp = -(*floatp);
          exp = tilep[k++] >> 4;
          if(tilep[k++] & 1) exp ^= 0x80000000;
        }

        /* replace NaN's with tileNum */
        if(isnan(*floatp))
          *floatp = (float) tilenum;

        /* perform floating point operation */
        *floatp = ldexpf(*floatp, exp);
        
        /* pre-scramble op */
        op ^= (uint32_t)tilep[j];
      }
      
      /* perform TILE_TRANSFORMS bit manipulations per row */
      for(t = 0; t < TILE_TRANSFORMS; t++) {
        /* determine tile byte offset and operation to use */
        op += (uint32_t)tilep[i + (t & 0x1f)];

        /* Original op selection by Ortis:
          	  for(int z = (h ^ i ^ t) % (HASHLEN >> 1);z<HASHLEN;z++)
        		op += hash[z];

        	Chris though it was too easy so he went for floating point arithmetic
        */

        switch(op & 7) {
          case 0: /* Swap the first and last bit in each byte. */
          {
            for(z = 0;z<HASHLEN;z++)
              tilep[i + z] ^= 0x81;
          }
            break;
          case 1: /* Swap bytes */
          {
            for(z = 0;z<HASHLENMID;z++)
            {
              bits = tilep[i + z];
              tilep[i + z] = tilep[i + HASHLENMID + z];
              tilep[i + HASHLENMID + z] = bits;
            }
          }
            break;
          case 2: /* Complement One all bytes */
          {
            for(z = 1; z < HASHLEN; z++)
              tilep[i + z] = ~tilep[i + z];
          }
            break;
          case 3: /* Alternate +1 and -1 on all bytes */
          {
            for(z = 0; z < HASHLEN; z++)
              tilep[i + z] += (z & 1 == 0) ? 1 : -1;
          }
            break;
          case 4: /* Alternate +t and -t on all bytes */
          {
            for(z = 0; z < HASHLEN; z++)
              tilep[i + z] += (z & 1 == 0) ? -t : t;
          }
            break;
          case 5: /* Replace every occurrence of h with H */
          {
            for(z = 0;z<HASHLEN;z++)
              if(tilep[i + z] == _104)
                tilep[i + z] = _72;
          }
            break;
          case 6: /* If byte a is > byte b, swap them. */
          {
            for(z = 0;z<HASHLENMID;z++)
              if(tilep[i + z] > tilep[i + HASHLENMID + z])
              {
                bits = tilep[i + z];
                tilep[i + z] = tilep[i + HASHLENMID + z];
                tilep[i + HASHLENMID + z] = bits;
              }
          }
            break;
          case 7 : /* XOR all bytes */
          {
            for(z = 1; z < HASHLEN; z++)
              tilep[i + z] ^= tilep[i + z - 1];
          }
            break;
        }
      }
      
      /* hash the result of the current tile's row to the next */
      if(j < TILE_LENGTH) {
        cuda_sha256_init(&ictx);
        cuda_sha256_update(&ictx, &tilep[i], HASHLEN);
        cuda_sha256_update(&ictx, (byte*)&tilenum, sizeof(uint32_t));
        cuda_sha256_final(&ictx, &tilep[j]);
      }
    }

 return tilep;
}

__global__ void cuda_find_peach(uint32_t threads, uint8_t *g_map, uint8_t *g_cache, 
                           int *g_found, uint8_t *g_seed) {

  const uint32_t thread = blockDim.x * blockIdx.x + threadIdx.x;
  cuda_SHA256_CTX ictx;
  uint32_t sm;
  uint8_t bt_hash[32], fhash[32], *tilep, *bp, n;
  uint8_t seed[16] = {0}, nonce[32] = {0};
  int i, j;

  
   if (thread <= threads) {
      /* Frame 1 -> Split 6 ways */
      if(thread < 32768) { /* Total Permutations, this frame: 32,768 ( 1 << 15 ) */
         seed[ 0] = Z_PREP[(thread & 3)];       // 2^2
         seed[ 1] = Z_TIMED[(thread >> 2) & 7]; // 2^3
         seed[ 2] = 1;
         seed[ 3] = 5;
         seed[ 4] = Z_NS[(thread >> 5) & 63];   // 2^6
         seed[ 5] = 1;
         seed[ 6] = Z_ING[(thread >> 11) & 15]; // 2^4
      } else
      if(thread < 49152) { /* Total Permutations, this frame: 16,384 ( 1 << 14 ) */
         seed[ 0] = Y_PREP[(thread & 1)];       // 2^1
         seed[ 1] = Z_TIMED[(thread >> 1) & 7]; // 2^3
         seed[ 2] = 1;
         seed[ 3] = 5;
         seed[ 4] = Z_NS[(thread >> 4) & 63];   // 2^6
         seed[ 5] = 1;
         seed[ 6] = Z_ING[(thread >> 10) & 15]; // 2^4
      } else
      if(thread < 65536) { /* Total Permutations, this frame: 16,384 ( 1 << 14 ) */
         seed[ 0] = Z_PREP[(thread & 3)];       // 2^2
         seed[ 1] = Z_TIMED[(thread >> 2) & 7]; // 2^3
         seed[ 2] = 1;
         seed[ 3] = 5;
         seed[ 4] = Z_NS[(thread >> 5) & 63];   // 2^6
         seed[ 5] = 1;
         seed[ 6] = Y_ING[(thread >> 11) & 7]; // 2^3
      } else
      if(thread < 73728) { /* Total Permutations, this frame: 8,192 ( 1 << 13 ) */
         seed[ 0] = Y_PREP[(thread & 1)];       // 2^1
         seed[ 1] = Z_TIMED[(thread >> 1) & 7]; // 2^3
         seed[ 2] = 1;
         seed[ 3] = 5;
         seed[ 4] = Z_NS[(thread >> 4) & 63];   // 2^6
         seed[ 5] = 1;
         seed[ 6] = Y_ING[(thread >> 10) & 7]; // 2^3
      } else
      if(thread < 81920) { /* Total Permutations, this frame: 16,384 ( 1 << 13 ) */
         seed[ 0] = Z_PREP[(thread & 3)];       // 2^2
         seed[ 1] = Z_TIMED[(thread >> 2) & 7]; // 2^3
         seed[ 2] = 1;
         seed[ 3] = 5;
         seed[ 4] = Z_NS[(thread >> 5) & 63];   // 2^6
         seed[ 5] = 1;
         seed[ 6] = X_ING[(thread >> 11) & 3]; // 2^2
      } else
      if(thread < 86016) { /* Total Permutations, this frame: 8,192 ( 1 << 12 ) */
         seed[ 0] = Y_PREP[(thread & 1)];       // 2^1
         seed[ 1] = Z_TIMED[(thread >> 1) & 7]; // 2^3
         seed[ 2] = 1;
         seed[ 3] = 5;
         seed[ 4] = Z_NS[(thread >> 4) & 63];   // 2^6
         seed[ 5] = 1;
         seed[ 6] = X_ING[(thread >> 10) & 3];  // 2^2
      } else
      /* END Frame 1 */
      /* Frame 2 -> Split 3 ways */
      if(thread <= 151552) { /* Total Permutations, this frame: 65,536 (1 << 16) */
         seed[ 0] = Z_TIME[(thread & 15)];      // 2^4
         seed[ 1] = Y_MASS[(thread >> 4) & 15]; // 2^4
         seed[ 2] = 1;
         seed[ 3] = Z_INF[(thread >> 8) & 15];  // 2^4
         seed[ 4] = 9;
         seed[ 5] = 2;
         seed[ 6] = 1;
         seed[ 7] = Z_AMB[(thread >> 12) & 15]; // 2^4
      } else
      if(thread <= 184320) { /* Total Permutations, this frame: 32,768 (1 << 15) */
         seed[ 0] = Z_TIME[(thread & 15)];      // 2^4
         seed[ 1] = X_MASS[(thread >> 3) & 7];  // 2^3
         seed[ 2] = 1;
         seed[ 3] = Z_INF[(thread >> 7) & 15];  // 2^4
         seed[ 4] = 9;
         seed[ 5] = 2;
         seed[ 6] = 1;
         seed[ 7] = Z_AMB[(thread >> 11) & 15]; // 2^4
      } else
      if(thread <= 200704) { /* Total Permutations, this frame: 16,384 (1 << 14) */
         seed[ 0] = Z_TIME[(thread & 15)];      // 2^4
         seed[ 1] = Z_MASS[(thread >> 2) & 3];  // 2^2
         seed[ 2] = 1;
         seed[ 3] = Z_INF[(thread >> 6) & 15];  // 2^4
         seed[ 4] = 9;
         seed[ 5] = 2;
         seed[ 6] = 1;
         seed[ 7] = Z_AMB[(thread >> 10) & 15]; // 2^4
      } else
      /* END Frame 2 */
      /* Frame 3 -> Split 2 ways */
      if(thread < 2297856) { /* Total Permutations, this frame: 2,097,152 ( 1 << 21 )*/
         seed[ 0] = Z_PREP[(thread & 3)];          // 2^2
         seed[ 1] = Z_TIMED[(thread >> 2) & 7];    // 2^3
         seed[ 2] = 1;
         seed[ 3] = Z_ADJ[(thread >> 5) & 63];     // 2^6
         seed[ 4] = Z_NPL[(thread >> 11) & 31];    // 2^5
         seed[ 5] = 1;
         seed[ 6] = Z_INGINF[(thread >> 16) & 31]; // 2^5
      } else
      if(thread < 3346432) { /* Total Permutations, this frame: 1,048,576 ( 1 << 20 )*/
         seed[ 0] = Y_PREP[(thread & 1)];          // 2^1
         seed[ 1] = Z_TIMED[(thread >> 1) & 7];    // 2^3
         seed[ 2] = 1;
         seed[ 3] = Z_ADJ[(thread >> 4) & 63];     // 2^6
         seed[ 4] = Z_NPL[(thread >> 10) & 31];    // 2^5
         seed[ 5] = 1;
         seed[ 6] = Z_INGINF[(thread >> 15) & 31]; // 2^5
      } else
      /* END Frame 3 */
      /* Frame 4 -> Split 6 ways */
      if(thread < 5443584) { /* Total Permutations, this frame: 2,097,152 ( 1 << 21 ) */
         seed[ 0] = 5;
         seed[ 1] = Z_NS[(thread & 63)];           // 2^6
         seed[ 2] = 1;
         seed[ 3] = Z_PREP[(thread >> 6) & 3];     // 2^2
         seed[ 4] = Z_TIMED[(thread >> 8) & 7];    // 2^3
         seed[ 5] = Z_MASS[(thread >> 11) & 15];   // 2^4
         seed[ 6] = 3;
         seed[ 7] = 1;
         seed[ 8] = Z_ADJ[(thread >> 15) & 63];    // 2^6
      } else
      if(thread < 6492160) { /* Total Permutations, this frame: 1,048,576 ( 1 << 20 ) */
         seed[ 0] = 5;
         seed[ 1] = Z_NS[(thread & 63)];           // 2^6
         seed[ 2] = 1;
         seed[ 3] = Y_PREP[(thread >> 6) & 1];     // 2^1
         seed[ 4] = Z_TIMED[(thread >> 7) & 7];    // 2^3
         seed[ 5] = Z_MASS[(thread >> 10) & 15];   // 2^4
         seed[ 6] = 3;
         seed[ 7] = 1;
         seed[ 8] = Z_ADJ[(thread >> 14) & 63];    // 2^6
      } else
      if(thread < 7540736) { /* Total Permutations, this frame: 1,048,576 ( 1 << 20 ) */
         seed[ 0] = 5;
         seed[ 1] = Z_NS[(thread & 63)];           // 2^6
         seed[ 2] = 1;
         seed[ 3] = Z_PREP[(thread >> 6) & 3];     // 2^2
         seed[ 4] = Z_TIMED[(thread >> 8) & 7];    // 2^3
         seed[ 5] = Y_MASS[(thread >> 11) & 7];    // 2^3
         seed[ 6] = 3;
         seed[ 7] = 1;
         seed[ 8] = Z_ADJ[(thread >> 14) & 63];    // 2^6
      } else
      if(thread < 8065024) { /* Total Permutations, this frame: 524,288 ( 1 << 19 ) */
         seed[ 0] = 5;
         seed[ 1] = Z_NS[(thread & 63)];           // 2^6
         seed[ 2] = 1;
         seed[ 3] = Y_PREP[(thread >> 6) & 1];     // 2^1
         seed[ 4] = Z_TIMED[(thread >> 7) & 7];    // 2^3
         seed[ 5] = Y_MASS[(thread >> 10) & 7];    // 2^3
         seed[ 6] = 3;
         seed[ 7] = 1;
         seed[ 8] = Z_ADJ[(thread >> 13) & 63];    // 2^6
      } else
      if(thread < 8589312) { /* Total Permutations, this frame: 524,288 ( 1 << 19 ) */
         seed[ 0] = 5;
         seed[ 1] = Z_NS[(thread & 63)];           // 2^6
         seed[ 2] = 1;
         seed[ 3] = Z_PREP[(thread >> 6) & 3];     // 2^2
         seed[ 4] = Z_TIMED[(thread >> 8) & 7];    // 2^3
         seed[ 5] = X_MASS[(thread >> 11) & 3];    // 2^2
         seed[ 6] = 3;
         seed[ 7] = 1;
         seed[ 8] = Z_ADJ[(thread >> 13) & 63];    // 2^6
      } else
      if(thread < 8851456) { /* Total Permutations, this frame: 262,144 ( 1 << 18 ) */
         seed[ 0] = 5;
         seed[ 1] = Z_NS[(thread & 63)];           // 2^6
         seed[ 2] = 1;
         seed[ 3] = Y_PREP[(thread >> 6) & 1];     // 2^1
         seed[ 4] = Z_TIMED[(thread >> 7) & 7];    // 2^3
         seed[ 5] = X_MASS[(thread >> 10) & 3];    // 2^2
         seed[ 6] = 3;
         seed[ 7] = 1;
         seed[ 8] = Z_ADJ[(thread >> 12) & 63];    // 2^6
      } else
      /* END Frame 4 */
      /* Frame 5 -> Split 6 ways */
      if(thread < 13045760) { /* Total Permutations, this frame: 4,194,304 ( 1 << 22 ) */
         seed[ 0] = Z_PREP[thread & 3];            // 2^2
         seed[ 1] = Z_ADJ[(thread >> 2) & 63];     // 2^6
         seed[ 2] = Z_MASS[(thread >> 8) & 15];    // 2^4
         seed[ 3] = 1;
         seed[ 4] = Z_NPL[(thread >> 12) & 31];    // 2^5
         seed[ 5] = 1;
         seed[ 6] = Z_INGINF[(thread >> 17) & 31]; // 2^5
      } else
      if(thread < 15142912) { /* Total Permutations, this frame: 2,097,152 ( 1 << 21 ) */
         seed[ 0] = Y_PREP[thread & 1];            // 2^1
         seed[ 1] = Z_ADJ[(thread >> 1) & 63];     // 2^6
         seed[ 2] = Z_MASS[(thread >> 7) & 15];    // 2^4
         seed[ 3] = 1;
         seed[ 4] = Z_NPL[(thread >> 11) & 31];    // 2^5
         seed[ 5] = 1;
         seed[ 6] = Z_INGINF[(thread >> 16) & 31]; // 2^5
      } else
      if(thread < 17240064) { /* Total Permutations, this frame: 2,097,152 ( 1 << 21 ) */
         seed[ 0] = Z_PREP[thread & 3];            // 2^2
         seed[ 1] = Z_ADJ[(thread >> 2) & 63];     // 2^6
         seed[ 2] = Y_MASS[(thread >> 8) & 7];     // 2^3
         seed[ 3] = 1;
         seed[ 4] = Z_NPL[(thread >> 11) & 31];    // 2^5
         seed[ 5] = 1;
         seed[ 6] = Z_INGINF[(thread >> 16) & 31]; // 2^5
      } else
      if(thread < 18288640) { /* Total Permutations, this frame: 1,048,576 ( 1 << 20 ) */
         seed[ 0] = Y_PREP[thread & 1];            // 2^1
         seed[ 1] = Z_ADJ[(thread >> 1) & 63];     // 2^6
         seed[ 2] = Y_MASS[(thread >> 7) & 7];     // 2^3
         seed[ 3] = 1;
         seed[ 4] = Z_NPL[(thread >> 10) & 31];    // 2^5
         seed[ 5] = 1;
         seed[ 6] = Z_INGINF[(thread >> 15) & 31]; // 2^5
      } else
      if(thread < 19337216) { /* Total Permutations, this frame: 1,048,576 ( 1 << 20 ) */
         seed[ 0] = Z_PREP[thread & 3];            // 2^2
         seed[ 1] = Z_ADJ[(thread >> 2) & 63];     // 2^6
         seed[ 2] = X_MASS[(thread >> 8) & 3];     // 2^2
         seed[ 3] = 1;
         seed[ 4] = Z_NPL[(thread >> 10) & 31];    // 2^5
         seed[ 5] = 1;
         seed[ 6] = Z_INGINF[(thread >> 15) & 31]; // 2^5
      } else
      if(thread < 19861504) { /* Total Permutations, this frame: 524,288 ( 1 << 19 ) */
         seed[ 0] = Y_PREP[thread & 1];            // 2^1
         seed[ 1] = Z_ADJ[(thread >> 1) & 63];     // 2^6
         seed[ 2] = X_MASS[(thread >> 7) & 3];     // 2^2
         seed[ 3] = 1;
         seed[ 4] = Z_NPL[(thread >> 9) & 31];     // 2^5
         seed[ 5] = 1;
         seed[ 6] = Z_INGINF[(thread >> 14) & 31]; // 2^5
      } else
      /* END Frame 5 */
      /* Frame 6 -> Split 6 ways */
      if(thread < 24055808) { /* Total Permutations, this frame: 4,194,304 ( 1 << 22 ) */
         seed[ 0] = Z_PREP[(thread & 3)];          // 2^2
         seed[ 1] = Z_MASS[(thread >> 2) & 15];    // 2^4
         seed[ 2] = 1;
         seed[ 3] = Z_ADJ[(thread >> 6) & 63];     // 2^6
         seed[ 4] = Z_NPL[(thread >> 12) & 31];    // 2^5
         seed[ 5] = 1;
         seed[ 6] = Z_INGINF[(thread >> 17) & 31]; // 2^5
      } else
      if(thread < 26152960) { /* Total Permutations, this frame: 2,097,152 ( 1 << 21 ) */
         seed[ 0] = Y_PREP[(thread & 1)];          // 2^1
         seed[ 1] = Z_MASS[(thread >> 1) & 15];    // 2^4
         seed[ 2] = 1;
         seed[ 3] = Z_ADJ[(thread >> 5) & 63];     // 2^6
         seed[ 4] = Z_NPL[(thread >> 11) & 31];    // 2^5
         seed[ 5] = 1;
         seed[ 6] = Z_INGINF[(thread >> 16) & 31]; // 2^5
      } else
      if(thread < 28250112) { /* Total Permutations, this frame: 2,097,152 ( 1 << 21 ) */
         seed[ 0] = Z_PREP[(thread & 3)];          // 2^2
         seed[ 1] = Y_MASS[(thread >> 2) & 7];     // 2^3
         seed[ 2] = 1;
         seed[ 3] = Z_ADJ[(thread >> 5) & 63];     // 2^6
         seed[ 4] = Z_NPL[(thread >> 11) & 31];    // 2^5
         seed[ 5] = 1;
         seed[ 6] = Z_INGINF[(thread >> 16) & 31]; // 2^5
      } else
      if(thread < 29298688) { /* Total Permutations, this frame: 1,048,576 ( 1 << 20 ) */
         seed[ 0] = Y_PREP[(thread & 1)];          // 2^1
         seed[ 1] = Y_MASS[(thread >> 1) & 7];     // 2^3
         seed[ 2] = 1;
         seed[ 3] = Z_ADJ[(thread >> 4) & 63];     // 2^6
         seed[ 4] = Z_NPL[(thread >> 10) & 31];    // 2^5
         seed[ 5] = 1;
         seed[ 6] = Z_INGINF[(thread >> 15) & 31]; // 2^5
      } else
      if(thread < 30347264) { /* Total Permutations, this frame: 1,048,576 ( 1 << 20 ) */
         seed[ 0] = Z_PREP[(thread & 3)];          // 2^2
         seed[ 1] = X_MASS[(thread >> 2) & 3];     // 2^2
         seed[ 2] = 1;
         seed[ 3] = Z_ADJ[(thread >> 5) & 63];     // 2^6
         seed[ 4] = Z_NPL[(thread >> 11) & 31];    // 2^5
         seed[ 5] = 1;
         seed[ 6] = Z_INGINF[(thread >> 16) & 31]; // 2^5
      } else
      if(thread < 30871552) { /* Total Permutations, this frame: 524,288 ( 1 << 19 ) */
         seed[ 0] = Y_PREP[(thread & 1)];          // 2^1
         seed[ 1] = X_MASS[(thread >> 1) & 3];     // 2^2
         seed[ 2] = 1;
         seed[ 3] = Z_ADJ[(thread >> 4) & 63];     // 2^6
         seed[ 4] = Z_NPL[(thread >> 10) & 31];    // 2^5
         seed[ 5] = 1;
         seed[ 6] = Z_INGINF[(thread >> 15) & 31]; // 2^5
      } else
      /* END Frame 6 */
      /* Frame 7 -> Split 9 ways */
      if(thread < 35065856) { /* Total Permutations, this frame: 4,194,304 ( 1 << 22 ) */
         seed[ 0] = Z_TIME[(thread & 15)];         // 2^4
         seed[ 1] = Z_AMB[(thread >> 4) & 15];     // 2^4
         seed[ 2] = 1;
         seed[ 3] = Z_ADJ[(thread >> 8) & 63];     // 2^6
         seed[ 4] = Z_MASS[(thread >> 14) & 15];   // 2^4
         seed[ 5] = 1;
         seed[ 6] = Z_ING[(thread >> 18) & 15];    // 2^4
      } else
      if(thread < 37163088) { /* Total Permutations, this frame: 2,097,152 ( 1 << 21 ) */
         seed[ 0] = Z_TIME[(thread & 15)];         // 2^4
         seed[ 1] = Z_AMB[(thread >> 4) & 15];     // 2^4
         seed[ 2] = 1;
         seed[ 3] = Z_ADJ[(thread >> 8) & 63];     // 2^6
         seed[ 4] = Y_MASS[(thread >> 14) & 7];    // 2^3
         seed[ 5] = 1;
         seed[ 6] = Z_ING[(thread >> 17) & 15];    // 2^4
      } else
      if(thread < 38211584) { /* Total Permutations, this frame: 1,048,576 ( 1 << 20 ) */
         seed[ 0] = Z_TIME[(thread & 15)];         // 2^4
         seed[ 1] = Z_AMB[(thread >> 4) & 15];     // 2^4
         seed[ 2] = 1;
         seed[ 3] = Z_ADJ[(thread >> 8) & 63];     // 2^6
         seed[ 4] = X_MASS[(thread >> 14) & 3];    // 2^2
         seed[ 5] = 1;
         seed[ 6] = Z_ING[(thread >> 16) & 15];    // 2^4
      } else
      if(thread < 40308736) { /* Total Permutations, this frame: 2,097,152 ( 1 << 21 ) */
         seed[ 0] = Z_TIME[(thread & 15)];         // 2^4
         seed[ 1] = Z_AMB[(thread >> 4) & 15];     // 2^4
         seed[ 2] = 1;
         seed[ 3] = Z_ADJ[(thread >> 8) & 63];     // 2^6
         seed[ 4] = Z_MASS[(thread >> 14) & 15];   // 2^4
         seed[ 5] = 1;
         seed[ 6] = Y_ING[(thread >> 18) & 7];     // 2^3
      } else
      if(thread < 41357312) { /* Total Permutations, this frame: 1,048,576 ( 1 << 20 ) */
         seed[ 0] = Z_TIME[(thread & 15)];         // 2^4
         seed[ 1] = Z_AMB[(thread >> 4) & 15];     // 2^4
         seed[ 2] = 1;
         seed[ 3] = Z_ADJ[(thread >> 8) & 63];     // 2^6
         seed[ 4] = Y_MASS[(thread >> 14) & 7];    // 2^3
         seed[ 5] = 1;
         seed[ 6] = Y_ING[(thread >> 17) & 7];     // 2^3
      } else
      if(thread < 41881600) { /* Total Permutations, this frame: 524,288 ( 1 << 19 ) */
         seed[ 0] = Z_TIME[(thread & 15)];         // 2^4
         seed[ 1] = Z_AMB[(thread >> 4) & 15];     // 2^4
         seed[ 2] = 1;
         seed[ 3] = Z_ADJ[(thread >> 8) & 63];     // 2^6
         seed[ 4] = X_MASS[(thread >> 14) & 3];    // 2^2
         seed[ 5] = 1;
         seed[ 6] = Y_ING[(thread >> 16) & 7];     // 2^3
      } else
      if(thread < 42405888) { /* Total Permutations, this frame: 524,288 ( 1 << 19  ) */
         seed[ 0] = Z_TIME[(thread & 15)];         // 2^4
         seed[ 1] = Z_AMB[(thread >> 4) & 15];     // 2^4
         seed[ 2] = 1;
         seed[ 3] = Z_ADJ[(thread >> 8) & 63];     // 2^6
         seed[ 4] = Z_MASS[(thread >> 14) & 15];   // 2^4
         seed[ 5] = 1;
         seed[ 6] = X_ING[(thread >> 18) & 1];     // 2^1
      } else
      if(thread < 42668032) { /* Total Permutations, this frame: 262,144 ( 1 << 18 ) */
         seed[ 0] = Z_TIME[(thread & 15)];         // 2^4
         seed[ 1] = Z_AMB[(thread >> 4) & 15];     // 2^4
         seed[ 2] = 1;
         seed[ 3] = Z_ADJ[(thread >> 8) & 63];     // 2^6
         seed[ 4] = Y_MASS[(thread >> 14) & 7];    // 2^3
         seed[ 5] = 1;
         seed[ 6] = X_ING[(thread >> 17) & 1];     // 2^1
      } else
      if(thread < 42799104) { /* Total Permutations, this frame: 131,072 ( 1 << 17 ) */
         seed[ 0] = Z_TIME[(thread & 15)];         // 2^4
         seed[ 1] = Z_AMB[(thread >> 4) & 15];     // 2^4
         seed[ 2] = 1;
         seed[ 3] = Z_ADJ[(thread >> 8) & 63];     // 2^6
         seed[ 4] = X_MASS[(thread >> 14) & 3];    // 2^2
         seed[ 5] = 1;
         seed[ 6] = X_ING[(thread >> 16) & 1];     // 2^1
      } else
      /* END Frame 7 */
      /* Frame 8 -> Split 2 ways */
      if(thread < 311234560) { /* Total Permutations, this frame: 268,435,456 ( 1 << 28 ) */
         seed[ 0] = Z_TIME[(thread & 15)];         // 2^4
         seed[ 1] = Z_AMB[(thread >> 4) & 15];     // 2^4
         seed[ 2] = 1;
         seed[ 3] = Z_PREP[(thread >> 8) & 3];     // 2^2
         seed[ 4] = 5;
         seed[ 5] = Z_ADJ[(thread >> 10) & 63];    // 2^6
         seed[ 6] = Z_NS[(thread >> 16) & 63];     // 2^6
         seed[ 7] = 3;
         seed[ 8] = 1;
         seed[ 9] = Z_INGADJ[(thread >> 22) & 63]; // 2^6
      } else
      if(thread < 445452288) { /* Total Permutations, this frame: 134,217,728 ( 1 << 27 ) */
         seed[ 0] = Z_TIME[(thread & 15)];         // 2^4
         seed[ 1] = Z_AMB[(thread >> 4) & 15];     // 2^4
         seed[ 2] = 1;
         seed[ 3] = Y_PREP[(thread >> 8) & 1];     // 2^1
         seed[ 4] = 5;
         seed[ 5] = Z_ADJ[(thread >> 9) & 63];     // 2^6
         seed[ 6] = Z_NS[(thread >> 15) & 63];     // 2^6
         seed[ 7] = 3;
         seed[ 8] = 1;
         seed[ 9] = Z_INGADJ[(thread >> 21) & 63]; // 2^6
      }

/* Below Two Frames are Valid, But Require 64-Bit Math: if extra entropy req'd.
   if( < thread <= ) { /* Total Permutations, this frame: 549,755,813,888
	seed[ 0] = Z_ING[(thread & 31)]; 
	seed[ 1] = Z_PREP[(thread << 5) & 7];
	seed[ 2] = Z_TIME[(thread << 8) & 15]; 
	seed[ 3] = Z_MASS[(thread << 12) & 31]; 
	seed[ 4] = 1;
        seed[ 5] = Z_MASS[(thread << 17) & 31]; 
	seed[ 6] = Z_ING[(thread << 22) & 31];  
	seed[ 7] = 3; 
	seed[ 8] = 1;
        seed[ 9] = 5; 
	seed[10] = Z_ADJ[(thread << 27) & 63];
	seed[11] = Z_NS[(thread << 33) & 63];
   }
   if( < thread <= ) { /* Total Permutations, this frame: 4,398,046,511,104
	seed[ 0] = Z_ING[(thread & 31)]; 
	seed[ 1] = Z_PREP[(thread << 5) & 7]; 
	seed[ 2] = 5; 
	seed[ 3] = Z_ADJ[(thread << 8) & 63]; 
	seed[ 4] = Z_NS[(thread << 14) & 63]; 
	seed[ 5] = 1;
        seed[ 6] = Z_MASS[(thread << 19) & 31]; 
	seed[ 7] = Z_ING[(thread << 24) & 31];  
	seed[ 8] = 3; 
	seed[ 9] = 1;
        seed[10] = 5; 
	seed[11] = Z_ADJ[(thread << 30) & 63]; 
	seed[12] = Z_NS[(thread << 36) & 63];
   }
End 64-bit Frames */
     
     /* store full nonce */
     #pragma unroll
     for (i = 0; i < 16; i++)
       nonce[i] = c_input32[i + 7];
     
     #pragma unroll
     for (i = 0; i < 16; i++)
       nonce[i+16] = seed[i];
     
     /*********************************************************/
     /* Hash 124 bytes of Block Trailer, including both seeds */
     /* Get the wizard to draw you a map to the princess!     */
     
     cuda_sha256_init(&ictx);

     /* update sha with the available block trailer data */
     cuda_sha256_update(&ictx, c_input32, 108);
     
     /* update sha with the second seed (16 bytes) */
     cuda_sha256_update(&ictx, seed, 16);
     
     /* finalise sha256 hash */
     cuda_sha256_final(&ictx, bt_hash);
     
for(i = 0; i < 32; i++) 
  printf(" %02X", bt_hash[i]);
printf("\n");
     
     /*****************************************************/
     /* Determine the final tile based on selected nonce  */
     /* Follow the wizard's map to find the princess!     */
     
     /* determine first tile index */
     sm = bt_hash[0];
     for(i = 1; i < HASHLEN; i++)
       sm *= bt_hash[i];
     
     sm %= MAP;

     /* get cached tile, or generate one if it doesn't exist */
     if(g_cache[sm])
       tilep = &g_map[sm * TILE_LENGTH];
     else
       tilep = cuda_gen_tile(sm, c_phash, g_map, g_cache);
     
     /* make <JUMP> tile jumps to find the final tile */
     for(j = 0; j < JUMP; j++) {
       /* determine next tile index */
       sm = cuda_next_index(sm, tilep, nonce);
       
       /* get cached tile, or generate one if it doesn't exist */
       if(g_cache[sm])
         tilep = &g_map[sm * TILE_LENGTH];
       else
         tilep = cuda_gen_tile(sm, c_phash, g_map, g_cache);
     }


     /****************************************************************/
     /* Check the hash of the final tile produces the desired result */
     /* Search the castle for the princess!                          */
     
     cuda_sha256_init(&ictx);
     cuda_sha256_update(&ictx, bt_hash, HASHLEN);
     cuda_sha256_update(&ictx, tilep, TILE_LENGTH);
     cuda_sha256_final(&ictx, fhash);
     
     for (bp = fhash, n = c_difficulty >> 5; n; n--)
       if (*bp++ != 0) return; /* Our princess is in another castle ! */
     if (__clz(*bp) >= (c_difficulty & 31)) {
       /* PRINCESS FOUND! */
       *g_found = 1;
       #pragma unroll
       for (i = 0; i < 16; i++)
         g_seed[i] = seed[i];
     }
   }
}



extern "C" {

typedef struct __peach_cuda_ctx {
    byte curr_seed[16], next_seed[16];
    char cp[256], next_cp[256];
    int *found, *d_found;
    uint8_t *seed, *d_seed, *d_cache;
    uint8_t *input, *d_map;
    hipStream_t stream;
} PeachCudaCTX;

PeachCudaCTX ctx[63];    /* Max 63 GPUs Supported */
uint32_t threads = 1;
dim3 grid(1);
dim3 block(1);
char nullcp = '\0';
byte *diff;
byte *phash;
byte bnum[8] = {0};
byte initGPU = 0;
int nGPU = 0;

int init_cuda_peach(byte difficulty, byte *prevhash, byte *blocknumber) {
    /* Obtain and check system GPU count */
    hipGetDeviceCount(&nGPU);
    if(nGPU<1 || nGPU>63) return nGPU;
    /* Allocate pinned host memory */
    hipHostMalloc(&diff, 1);
    hipHostMalloc(&phash, 32);
    /* Copy immediate block data to pinned memory */
    memcpy(diff, &difficulty, 1);
    memcpy(phash, prevhash, 32);

    int i = 0;
    for ( ; i<nGPU; i++) {
        hipSetDevice(i);
        /* Create Stream */
        hipStreamCreate(&ctx[i].stream);
        /* Allocate device memory */
        hipMalloc(&ctx[i].d_found, 4);
        hipMalloc(&ctx[i].d_seed, 16);
        /* Allocate associated device-host memory */
        hipHostMalloc(&ctx[i].found, 4);
        hipHostMalloc(&ctx[i].seed, 16);
        hipHostMalloc(&ctx[i].input, 108);
        /* Copy immediate block data to device memory */
        hipMemcpyToSymbolAsync(HIP_SYMBOL(c_difficulty), diff, 1, 0,
                                hipMemcpyHostToDevice, ctx[i].stream);
        hipMemcpyToSymbolAsync(HIP_SYMBOL(c_phash), phash, 32, 0,
                                hipMemcpyHostToDevice, ctx[i].stream);
        /* Set remaining device memory */
        hipMemsetAsync(ctx[i].d_found, 0, 4, ctx[i].stream);
        hipMemsetAsync(ctx[i].d_seed, 0, 16, ctx[i].stream);
        /* Set initial round variables */
        ctx[i].next_cp[0] = nullcp;
        /* If first init, setup map and cache */
        if(initGPU == 0) {
            hipMalloc(&ctx[i].d_map, MAP_LENGTH);
            hipMalloc(&ctx[i].d_cache, MAP);
            initGPU = 1;
        }
        /* Wipe cache if new block */
        if(initGPU && memcmp(blocknumber, bnum, 8) != 0) {
            hipMemsetAsync(ctx[i].d_cache, 0, MAP, ctx[i].stream);
            memcpy(bnum, blocknumber, 8);
        }
    }

    return nGPU;
}

void free_cuda_peach() {
    /* Free pinned host memory */
    hipHostFree(diff);
    hipHostFree(phash);

    int i = 0;
    for ( ; i<nGPU; i++) {
        hipSetDevice(i);
        /* Destroy Stream */
        hipStreamDestroy(ctx[i].stream);
        /* Free device memory */
        hipFree(ctx[i].d_found);
        hipFree(ctx[i].d_seed);
        /* Free associated device-host memory */
        hipHostFree(ctx[i].found);
        hipHostFree(ctx[i].seed);
        hipHostFree(ctx[i].input);
    }
}

extern byte *trigg_gen(byte *in);
extern char *trigg_expand2(byte *in, byte *out);

__host__ char *cuda_peach(byte *bt, char *haiku, uint32_t *hps, byte *runflag)
{
    int i;
    uint64_t nHaiku;
    time_t start = time(NULL);
    for(haiku = NULL; *runflag && haiku == NULL; ) {
        for (i=0; i<nGPU; i++) {
            /* Prepare next seed for GPU... */
            if(ctx[i].next_cp[0] == nullcp) {
                /* ... generate first GPU seed (and expand as Haiku) */
                trigg_gen(ctx[i].next_seed);
                trigg_expand2(ctx[i].next_seed, (byte*)ctx[i].next_cp);

                /* ... and prepare round data */
                memcpy(ctx[i].input, bt, 108);
            }
            /* Check if GPU has finished */
            CudaCheckError();
            hipSetDevice(i);
            if(hipStreamQuery(ctx[i].stream) == hipSuccess) {
                hipMemcpy(ctx[i].found, ctx[i].d_found, 4, hipMemcpyDeviceToHost);
                if(*ctx[i].found==1) { /* SOLVED A BLOCK! */
                    hipMemcpy(ctx[i].seed, ctx[i].d_seed, 16, hipMemcpyDeviceToHost);
                    memcpy(bt + 92, ctx[i].curr_seed, 16);
                    memcpy(bt + 92 + 16, ctx[i].seed, 16);
                    haiku = ctx[i].cp;
                    break;
                }
                /* Send new GPU round Data */
                hipMemcpyToSymbolAsync(HIP_SYMBOL(c_input32), ctx[i].input, 108, 0,
                                        hipMemcpyHostToDevice, ctx[i].stream);
                /* Start GPU round */
                cuda_find_peach<<<grid, block, 0, ctx[i].stream>>>(threads,
                ctx[i].d_map, ctx[i].d_cache, ctx[i].d_found, ctx[i].d_seed);

                /* Add to haiku count */
                nHaiku += threads;

                /* Store round vars aside for checks next loop */
                memcpy(ctx[i].curr_seed,ctx[i].next_seed,16);
                strcpy(ctx[i].cp,ctx[i].next_cp);
                ctx[i].next_cp[0] = nullcp;
            } else continue;  /* Waiting on GPU ... */
        }
    }
    
    start = time(NULL) - start;
    if(start == 0) start = 1;
    *hps = (uint32_t)(nHaiku / (uint64_t)start);
         
    return haiku;
}


}
