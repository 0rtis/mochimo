#include "hip/hip_runtime.h"
/*
 * cuda_trigg.cu  Multi-GPU CUDA Mining
 *
 * Copyright (c) 2019 by Adequate Systems, LLC.  All Rights Reserved.
 * See LICENSE.PDF   **** NO WARRANTY ****
 *
 * Date: 10 August 2018
 * Revision: 31
 */

#include <stdio.h>
#include <stdint.h>
#include <inttypes.h>
#include <unistd.h>
#include <hip/hip_runtime.h>

#include "../../config.h"
#include "peach.h"
#include "nighthash.cu"

__constant__ static uint8_t __align__(8) c_phash[32];
__constant__ static uint8_t __align__(8) c_input32[108];
__constant__ static uint8_t __align__(8) c_difficulty;
__constant__ static int Z_PREP[4] = {12,13,14,15};
__constant__ static int Y_PREP[2] = {16,17};
__constant__ static int Z_ING[16] = {18,19,20,21,22,23,24,25,26,27,28,29,30,31,32,33};
__constant__ static int Y_ING[8]  = {34,35,36,37,38,39,40,41};
__constant__ static int X_ING[2]  = {42,43};
__constant__ static int Z_INF[16] = {44,45,46,47,48,50,51,52,53,54,55,56,57,58,59,60};
__constant__ static int Z_ADJ[64] =
   {61,62,63,64,65,66,67,68,69,70,71,72,73,74,75,76,77,78,79,80,81,82,83,84,85,86,87,
    88,89,90,91,92,94,95,96,97,98,99,100,101,102,103,104,105,107,108,109,110,112,114,
    115,116,117,118,119,120,121,122,123,124,125,126,127,128};
__constant__ static int Z_AMB[16] =
   {77,94,95,96,126,214,217,218,220,222,223,224,225,226,227,228};
__constant__ static int Z_TIMED[8] = {84,243,249,250,251,252,253,255};
__constant__ static int Z_NS[64] =
   {129,130,131,132,133,134,135,136,137,138,145,149,154,155,156,157,177,178,179,180,
    182,183,184,185,186,187,188,189,190,191,192,193,194,196,197,198,199,200,201,202,
    203,204,205,206,207,208,209,210,211,212,213,241,244,245,246,247,248,249,250,251,
    252,253,254,255};
__constant__ static int Z_NPL[32] =
   {139,140,141,142,143,144,146,147,148,150,151,153,158,159,160,161,162,163,164,165,
    166,167,168,169,170,171,172,173,174,175,176,181};
__constant__ static int Z_MASS[16] =
   {214,215,216,217,218,219,220,221,222,223,224,225,226,227,228,229};
__constant__ static int Y_MASS[8] = {230,231,232,233,234,235,236,237};
__constant__ static int X_MASS[4] = {238,239,240,242};
__constant__ static int Z_INGINF[32] =
   {18,19,20,21,22,25,26,27,28,29,30,36,37,38,39,40,41,42,44,46,47,48,49,51,52,53,54,
    55,56,57,58,59};
__constant__ static int Z_TIME[16] =
   {82,83,84,85,86,87,88,243,249,250,251,252,253,254,255,253};
__constant__ static int Z_INGADJ[64] =
   {18,19,20,21,22,23,24,25,26,27,28,29,30,31,32,33,34,35,36,37,38,39,40,41,42,43,23,
    24,31,32,33,34,61,62,63,64,65,66,67,68,69,70,71,72,73,74,75,76,77,78,79,80,81,82,
    83,84,85,86,87,88,89,90,91,92};


inline void cudaCheckError( const char *msg, uint32_t gpu, const char *file)
{
   hipError_t err = hipGetLastError();
   if(hipSuccess != err) {
      fprintf(stderr, "%s Error (#%d) in %s: %s\n",
              msg, gpu, file, hipGetErrorString(err));
      exit(-1);
   }
}


__device__ uint32_t cuda_next_index(uint32_t index, uint8_t *g_map, uint8_t *nonce)
{
   CUDA_NIGHTHASH_CTX nighthash;
   byte seed[HASHLEN + 4 + TILE_LENGTH];
   byte hash[HASHLEN];
   int i, seedlen;

   /* Create nighthash seed for this index on the map */
   seedlen = HASHLEN + 4 + TILE_LENGTH;
   memcpy(seed, nonce, HASHLEN);
   memcpy(seed + HASHLEN, (byte *) &index, 4);
   memcpy(seed + HASHLEN + 4, &g_map[index * TILE_LENGTH], TILE_LENGTH);
   
   /* Setup nighthash the seed, NO TRANSFORM */
   cuda_nighthash_init(&nighthash, seed, seedlen, index, 0);

   /* Update nighthash with the seed data */
   cuda_nighthash_update(&nighthash, seed, seedlen);

   /* Finalize nighthash into the first 32 byte chunk of the tile */
   cuda_nighthash_final(&nighthash, hash);

   /* Convert 32-byte Hash Value Into 8x 32-bit Unsigned Integer */
   for(i = 0, index = 0; i < 8; i++)
      index += ((uint32_t *) &hash)[i];

   return index % MAP;
}


__device__ void cuda_gen_tile(uint32_t index, uint8_t *phash, uint8_t *g_map)
{
   CUDA_NIGHTHASH_CTX nighthash;
   byte seed[4 + HASHLEN];
   byte *tilep;
   int i, j, seedlen;

   /* Set map pointer */
   tilep = &g_map[index * TILE_LENGTH];

   /* Create nighthash seed for this index on the map */
   seedlen = 4 + HASHLEN;
   memcpy(seed, (byte *) &index, 4);
   memcpy(seed + 4, phash, HASHLEN);

   /* Setup nighthash with a transform of the seed */
   cuda_nighthash_init(&nighthash, seed, seedlen, index, 1);

   /* Update nighthash with the seed data */
   cuda_nighthash_update(&nighthash, seed, seedlen);

   /* Finalize nighthash into the first 32 byte chunk of the tile */
   cuda_nighthash_final(&nighthash, tilep);

   /* Begin constructing the full tile */
   for(i = 0; i < TILE_LENGTH; i += HASHLEN) { /* For each tile row */
      /* Set next row's pointer location */
      j = i + HASHLEN;

      /* Hash the current row to the next, if not at the end */
      if(j < TILE_LENGTH) {
         /* Setup nighthash with a transform of the current row */
         cuda_nighthash_init(&nighthash, &tilep[i], HASHLEN, index, 1);

         /* Update nighthash with the seed data and tile index */
         cuda_nighthash_update(&nighthash, &tilep[i], HASHLEN);
         cuda_nighthash_update(&nighthash, (byte *) &index, 4);

         /* Finalize nighthash into the first 32 byte chunk of the tile */
         cuda_nighthash_final(&nighthash, &tilep[j]);
      }
   }
}


__global__ void cuda_build_map(uint32_t g_cache, uint8_t *g_map)
{
    const uint32_t thread = blockDim.x * blockIdx.x + threadIdx.x;
    
    if (thread < g_cache && thread < MAP) {
     
     /*****************************************************/
     /* Determine the final tile based on selected nonce. */
     /* Toadstool, get possible locations of the princess */
        
        cuda_gen_tile(thread, c_phash, g_map);
       
   }
}


__global__ void cuda_find_peach(uint32_t threads, int g_cache, uint8_t *g_map, 
                           int *g_found, uint8_t *g_seed)
{
  const uint32_t thread = blockDim.x * blockIdx.x + threadIdx.x;
  CUDA_SHA256_CTX ictx;
  uint32_t sm, sma[9];
  uint8_t bt_hash[32], fhash[32];
  uint8_t seed[16] = {0}, nonce[32] = {0};
  int i, j, n, x;

  
   if (thread <= threads) {
      /* Frame 1 -> Split 6 ways */
      if(thread < 32768) { /* Total Permutations, this frame: 32,768 ( 1 << 15 ) */
         seed[ 0] = Z_PREP[(thread & 3)];       // 2^2
         seed[ 1] = Z_TIMED[(thread >> 2) & 7]; // 2^3
         seed[ 2] = 1;
         seed[ 3] = 5;
         seed[ 4] = Z_NS[(thread >> 5) & 63];   // 2^6
         seed[ 5] = 1;
         seed[ 6] = Z_ING[(thread >> 11) & 15]; // 2^4
      } else
      if(thread < 49152) { /* Total Permutations, this frame: 16,384 ( 1 << 14 ) */
         seed[ 0] = Y_PREP[(thread & 1)];       // 2^1
         seed[ 1] = Z_TIMED[(thread >> 1) & 7]; // 2^3
         seed[ 2] = 1;
         seed[ 3] = 5;
         seed[ 4] = Z_NS[(thread >> 4) & 63];   // 2^6
         seed[ 5] = 1;
         seed[ 6] = Z_ING[(thread >> 10) & 15]; // 2^4
      } else
      if(thread < 65536) { /* Total Permutations, this frame: 16,384 ( 1 << 14 ) */
         seed[ 0] = Z_PREP[(thread & 3)];       // 2^2
         seed[ 1] = Z_TIMED[(thread >> 2) & 7]; // 2^3
         seed[ 2] = 1;
         seed[ 3] = 5;
         seed[ 4] = Z_NS[(thread >> 5) & 63];   // 2^6
         seed[ 5] = 1;
         seed[ 6] = Y_ING[(thread >> 11) & 7]; // 2^3
      } else
      if(thread < 73728) { /* Total Permutations, this frame: 8,192 ( 1 << 13 ) */
         seed[ 0] = Y_PREP[(thread & 1)];       // 2^1
         seed[ 1] = Z_TIMED[(thread >> 1) & 7]; // 2^3
         seed[ 2] = 1;
         seed[ 3] = 5;
         seed[ 4] = Z_NS[(thread >> 4) & 63];   // 2^6
         seed[ 5] = 1;
         seed[ 6] = Y_ING[(thread >> 10) & 7]; // 2^3
      } else
      if(thread < 81920) { /* Total Permutations, this frame: 16,384 ( 1 << 13 ) */
         seed[ 0] = Z_PREP[(thread & 3)];       // 2^2
         seed[ 1] = Z_TIMED[(thread >> 2) & 7]; // 2^3
         seed[ 2] = 1;
         seed[ 3] = 5;
         seed[ 4] = Z_NS[(thread >> 5) & 63];   // 2^6
         seed[ 5] = 1;
         seed[ 6] = X_ING[(thread >> 11) & 3]; // 2^2
      } else
      if(thread < 86016) { /* Total Permutations, this frame: 8,192 ( 1 << 12 ) */
         seed[ 0] = Y_PREP[(thread & 1)];       // 2^1
         seed[ 1] = Z_TIMED[(thread >> 1) & 7]; // 2^3
         seed[ 2] = 1;
         seed[ 3] = 5;
         seed[ 4] = Z_NS[(thread >> 4) & 63];   // 2^6
         seed[ 5] = 1;
         seed[ 6] = X_ING[(thread >> 10) & 3];  // 2^2
      } else
      /* END Frame 1 */
      /* Frame 2 -> Split 3 ways */
      if(thread <= 151552) { /* Total Permutations, this frame: 65,536 (1 << 16) */
         seed[ 0] = Z_TIME[(thread & 15)];      // 2^4
         seed[ 1] = Y_MASS[(thread >> 4) & 15]; // 2^4
         seed[ 2] = 1;
         seed[ 3] = Z_INF[(thread >> 8) & 15];  // 2^4
         seed[ 4] = 9;
         seed[ 5] = 2;
         seed[ 6] = 1;
         seed[ 7] = Z_AMB[(thread >> 12) & 15]; // 2^4
      } else
      if(thread <= 184320) { /* Total Permutations, this frame: 32,768 (1 << 15) */
         seed[ 0] = Z_TIME[(thread & 15)];      // 2^4
         seed[ 1] = X_MASS[(thread >> 3) & 7];  // 2^3
         seed[ 2] = 1;
         seed[ 3] = Z_INF[(thread >> 7) & 15];  // 2^4
         seed[ 4] = 9;
         seed[ 5] = 2;
         seed[ 6] = 1;
         seed[ 7] = Z_AMB[(thread >> 11) & 15]; // 2^4
      } else
      if(thread <= 200704) { /* Total Permutations, this frame: 16,384 (1 << 14) */
         seed[ 0] = Z_TIME[(thread & 15)];      // 2^4
         seed[ 1] = Z_MASS[(thread >> 2) & 3];  // 2^2
         seed[ 2] = 1;
         seed[ 3] = Z_INF[(thread >> 6) & 15];  // 2^4
         seed[ 4] = 9;
         seed[ 5] = 2;
         seed[ 6] = 1;
         seed[ 7] = Z_AMB[(thread >> 10) & 15]; // 2^4
      } else
      /* END Frame 2 */
      /* Frame 3 -> Split 2 ways */
      if(thread < 2297856) { /* Total Permutations, this frame: 2,097,152 ( 1 << 21 )*/
         seed[ 0] = Z_PREP[(thread & 3)];          // 2^2
         seed[ 1] = Z_TIMED[(thread >> 2) & 7];    // 2^3
         seed[ 2] = 1;
         seed[ 3] = Z_ADJ[(thread >> 5) & 63];     // 2^6
         seed[ 4] = Z_NPL[(thread >> 11) & 31];    // 2^5
         seed[ 5] = 1;
         seed[ 6] = Z_INGINF[(thread >> 16) & 31]; // 2^5
      } else
      if(thread < 3346432) { /* Total Permutations, this frame: 1,048,576 ( 1 << 20 )*/
         seed[ 0] = Y_PREP[(thread & 1)];          // 2^1
         seed[ 1] = Z_TIMED[(thread >> 1) & 7];    // 2^3
         seed[ 2] = 1;
         seed[ 3] = Z_ADJ[(thread >> 4) & 63];     // 2^6
         seed[ 4] = Z_NPL[(thread >> 10) & 31];    // 2^5
         seed[ 5] = 1;
         seed[ 6] = Z_INGINF[(thread >> 15) & 31]; // 2^5
      } else
      /* END Frame 3 */
      /* Frame 4 -> Split 6 ways */
      if(thread < 5443584) { /* Total Permutations, this frame: 2,097,152 ( 1 << 21 ) */
         seed[ 0] = 5;
         seed[ 1] = Z_NS[(thread & 63)];           // 2^6
         seed[ 2] = 1;
         seed[ 3] = Z_PREP[(thread >> 6) & 3];     // 2^2
         seed[ 4] = Z_TIMED[(thread >> 8) & 7];    // 2^3
         seed[ 5] = Z_MASS[(thread >> 11) & 15];   // 2^4
         seed[ 6] = 3;
         seed[ 7] = 1;
         seed[ 8] = Z_ADJ[(thread >> 15) & 63];    // 2^6
      } else
      if(thread < 6492160) { /* Total Permutations, this frame: 1,048,576 ( 1 << 20 ) */
         seed[ 0] = 5;
         seed[ 1] = Z_NS[(thread & 63)];           // 2^6
         seed[ 2] = 1;
         seed[ 3] = Y_PREP[(thread >> 6) & 1];     // 2^1
         seed[ 4] = Z_TIMED[(thread >> 7) & 7];    // 2^3
         seed[ 5] = Z_MASS[(thread >> 10) & 15];   // 2^4
         seed[ 6] = 3;
         seed[ 7] = 1;
         seed[ 8] = Z_ADJ[(thread >> 14) & 63];    // 2^6
      } else
      if(thread < 7540736) { /* Total Permutations, this frame: 1,048,576 ( 1 << 20 ) */
         seed[ 0] = 5;
         seed[ 1] = Z_NS[(thread & 63)];           // 2^6
         seed[ 2] = 1;
         seed[ 3] = Z_PREP[(thread >> 6) & 3];     // 2^2
         seed[ 4] = Z_TIMED[(thread >> 8) & 7];    // 2^3
         seed[ 5] = Y_MASS[(thread >> 11) & 7];    // 2^3
         seed[ 6] = 3;
         seed[ 7] = 1;
         seed[ 8] = Z_ADJ[(thread >> 14) & 63];    // 2^6
      } else
      if(thread < 8065024) { /* Total Permutations, this frame: 524,288 ( 1 << 19 ) */
         seed[ 0] = 5;
         seed[ 1] = Z_NS[(thread & 63)];           // 2^6
         seed[ 2] = 1;
         seed[ 3] = Y_PREP[(thread >> 6) & 1];     // 2^1
         seed[ 4] = Z_TIMED[(thread >> 7) & 7];    // 2^3
         seed[ 5] = Y_MASS[(thread >> 10) & 7];    // 2^3
         seed[ 6] = 3;
         seed[ 7] = 1;
         seed[ 8] = Z_ADJ[(thread >> 13) & 63];    // 2^6
      } else
      if(thread < 8589312) { /* Total Permutations, this frame: 524,288 ( 1 << 19 ) */
         seed[ 0] = 5;
         seed[ 1] = Z_NS[(thread & 63)];           // 2^6
         seed[ 2] = 1;
         seed[ 3] = Z_PREP[(thread >> 6) & 3];     // 2^2
         seed[ 4] = Z_TIMED[(thread >> 8) & 7];    // 2^3
         seed[ 5] = X_MASS[(thread >> 11) & 3];    // 2^2
         seed[ 6] = 3;
         seed[ 7] = 1;
         seed[ 8] = Z_ADJ[(thread >> 13) & 63];    // 2^6
      } else
      if(thread < 8851456) { /* Total Permutations, this frame: 262,144 ( 1 << 18 ) */
         seed[ 0] = 5;
         seed[ 1] = Z_NS[(thread & 63)];           // 2^6
         seed[ 2] = 1;
         seed[ 3] = Y_PREP[(thread >> 6) & 1];     // 2^1
         seed[ 4] = Z_TIMED[(thread >> 7) & 7];    // 2^3
         seed[ 5] = X_MASS[(thread >> 10) & 3];    // 2^2
         seed[ 6] = 3;
         seed[ 7] = 1;
         seed[ 8] = Z_ADJ[(thread >> 12) & 63];    // 2^6
      } else
      /* END Frame 4 */
      /* Frame 5 -> Split 6 ways */
      if(thread < 13045760) { /* Total Permutations, this frame: 4,194,304 ( 1 << 22 ) */
         seed[ 0] = Z_PREP[thread & 3];            // 2^2
         seed[ 1] = Z_ADJ[(thread >> 2) & 63];     // 2^6
         seed[ 2] = Z_MASS[(thread >> 8) & 15];    // 2^4
         seed[ 3] = 1;
         seed[ 4] = Z_NPL[(thread >> 12) & 31];    // 2^5
         seed[ 5] = 1;
         seed[ 6] = Z_INGINF[(thread >> 17) & 31]; // 2^5
      } else
      if(thread < 15142912) { /* Total Permutations, this frame: 2,097,152 ( 1 << 21 ) */
         seed[ 0] = Y_PREP[thread & 1];            // 2^1
         seed[ 1] = Z_ADJ[(thread >> 1) & 63];     // 2^6
         seed[ 2] = Z_MASS[(thread >> 7) & 15];    // 2^4
         seed[ 3] = 1;
         seed[ 4] = Z_NPL[(thread >> 11) & 31];    // 2^5
         seed[ 5] = 1;
         seed[ 6] = Z_INGINF[(thread >> 16) & 31]; // 2^5
      } else
      if(thread < 17240064) { /* Total Permutations, this frame: 2,097,152 ( 1 << 21 ) */
         seed[ 0] = Z_PREP[thread & 3];            // 2^2
         seed[ 1] = Z_ADJ[(thread >> 2) & 63];     // 2^6
         seed[ 2] = Y_MASS[(thread >> 8) & 7];     // 2^3
         seed[ 3] = 1;
         seed[ 4] = Z_NPL[(thread >> 11) & 31];    // 2^5
         seed[ 5] = 1;
         seed[ 6] = Z_INGINF[(thread >> 16) & 31]; // 2^5
      } else
      if(thread < 18288640) { /* Total Permutations, this frame: 1,048,576 ( 1 << 20 ) */
         seed[ 0] = Y_PREP[thread & 1];            // 2^1
         seed[ 1] = Z_ADJ[(thread >> 1) & 63];     // 2^6
         seed[ 2] = Y_MASS[(thread >> 7) & 7];     // 2^3
         seed[ 3] = 1;
         seed[ 4] = Z_NPL[(thread >> 10) & 31];    // 2^5
         seed[ 5] = 1;
         seed[ 6] = Z_INGINF[(thread >> 15) & 31]; // 2^5
      } else
      if(thread < 19337216) { /* Total Permutations, this frame: 1,048,576 ( 1 << 20 ) */
         seed[ 0] = Z_PREP[thread & 3];            // 2^2
         seed[ 1] = Z_ADJ[(thread >> 2) & 63];     // 2^6
         seed[ 2] = X_MASS[(thread >> 8) & 3];     // 2^2
         seed[ 3] = 1;
         seed[ 4] = Z_NPL[(thread >> 10) & 31];    // 2^5
         seed[ 5] = 1;
         seed[ 6] = Z_INGINF[(thread >> 15) & 31]; // 2^5
      } else
      if(thread < 19861504) { /* Total Permutations, this frame: 524,288 ( 1 << 19 ) */
         seed[ 0] = Y_PREP[thread & 1];            // 2^1
         seed[ 1] = Z_ADJ[(thread >> 1) & 63];     // 2^6
         seed[ 2] = X_MASS[(thread >> 7) & 3];     // 2^2
         seed[ 3] = 1;
         seed[ 4] = Z_NPL[(thread >> 9) & 31];     // 2^5
         seed[ 5] = 1;
         seed[ 6] = Z_INGINF[(thread >> 14) & 31]; // 2^5
      } else
      /* END Frame 5 */
      /* Frame 6 -> Split 6 ways */
      if(thread < 24055808) { /* Total Permutations, this frame: 4,194,304 ( 1 << 22 ) */
         seed[ 0] = Z_PREP[(thread & 3)];          // 2^2
         seed[ 1] = Z_MASS[(thread >> 2) & 15];    // 2^4
         seed[ 2] = 1;
         seed[ 3] = Z_ADJ[(thread >> 6) & 63];     // 2^6
         seed[ 4] = Z_NPL[(thread >> 12) & 31];    // 2^5
         seed[ 5] = 1;
         seed[ 6] = Z_INGINF[(thread >> 17) & 31]; // 2^5
      } else
      if(thread < 26152960) { /* Total Permutations, this frame: 2,097,152 ( 1 << 21 ) */
         seed[ 0] = Y_PREP[(thread & 1)];          // 2^1
         seed[ 1] = Z_MASS[(thread >> 1) & 15];    // 2^4
         seed[ 2] = 1;
         seed[ 3] = Z_ADJ[(thread >> 5) & 63];     // 2^6
         seed[ 4] = Z_NPL[(thread >> 11) & 31];    // 2^5
         seed[ 5] = 1;
         seed[ 6] = Z_INGINF[(thread >> 16) & 31]; // 2^5
      } else
      if(thread < 28250112) { /* Total Permutations, this frame: 2,097,152 ( 1 << 21 ) */
         seed[ 0] = Z_PREP[(thread & 3)];          // 2^2
         seed[ 1] = Y_MASS[(thread >> 2) & 7];     // 2^3
         seed[ 2] = 1;
         seed[ 3] = Z_ADJ[(thread >> 5) & 63];     // 2^6
         seed[ 4] = Z_NPL[(thread >> 11) & 31];    // 2^5
         seed[ 5] = 1;
         seed[ 6] = Z_INGINF[(thread >> 16) & 31]; // 2^5
      } else
      if(thread < 29298688) { /* Total Permutations, this frame: 1,048,576 ( 1 << 20 ) */
         seed[ 0] = Y_PREP[(thread & 1)];          // 2^1
         seed[ 1] = Y_MASS[(thread >> 1) & 7];     // 2^3
         seed[ 2] = 1;
         seed[ 3] = Z_ADJ[(thread >> 4) & 63];     // 2^6
         seed[ 4] = Z_NPL[(thread >> 10) & 31];    // 2^5
         seed[ 5] = 1;
         seed[ 6] = Z_INGINF[(thread >> 15) & 31]; // 2^5
      } else
      if(thread < 30347264) { /* Total Permutations, this frame: 1,048,576 ( 1 << 20 ) */
         seed[ 0] = Z_PREP[(thread & 3)];          // 2^2
         seed[ 1] = X_MASS[(thread >> 2) & 3];     // 2^2
         seed[ 2] = 1;
         seed[ 3] = Z_ADJ[(thread >> 5) & 63];     // 2^6
         seed[ 4] = Z_NPL[(thread >> 11) & 31];    // 2^5
         seed[ 5] = 1;
         seed[ 6] = Z_INGINF[(thread >> 16) & 31]; // 2^5
      } else
      if(thread < 30871552) { /* Total Permutations, this frame: 524,288 ( 1 << 19 ) */
         seed[ 0] = Y_PREP[(thread & 1)];          // 2^1
         seed[ 1] = X_MASS[(thread >> 1) & 3];     // 2^2
         seed[ 2] = 1;
         seed[ 3] = Z_ADJ[(thread >> 4) & 63];     // 2^6
         seed[ 4] = Z_NPL[(thread >> 10) & 31];    // 2^5
         seed[ 5] = 1;
         seed[ 6] = Z_INGINF[(thread >> 15) & 31]; // 2^5
      } else
      /* END Frame 6 */
      /* Frame 7 -> Split 9 ways */
      if(thread < 35065856) { /* Total Permutations, this frame: 4,194,304 ( 1 << 22 ) */
         seed[ 0] = Z_TIME[(thread & 15)];         // 2^4
         seed[ 1] = Z_AMB[(thread >> 4) & 15];     // 2^4
         seed[ 2] = 1;
         seed[ 3] = Z_ADJ[(thread >> 8) & 63];     // 2^6
         seed[ 4] = Z_MASS[(thread >> 14) & 15];   // 2^4
         seed[ 5] = 1;
         seed[ 6] = Z_ING[(thread >> 18) & 15];    // 2^4
      } else
      if(thread < 37163088) { /* Total Permutations, this frame: 2,097,152 ( 1 << 21 ) */
         seed[ 0] = Z_TIME[(thread & 15)];         // 2^4
         seed[ 1] = Z_AMB[(thread >> 4) & 15];     // 2^4
         seed[ 2] = 1;
         seed[ 3] = Z_ADJ[(thread >> 8) & 63];     // 2^6
         seed[ 4] = Y_MASS[(thread >> 14) & 7];    // 2^3
         seed[ 5] = 1;
         seed[ 6] = Z_ING[(thread >> 17) & 15];    // 2^4
      } else
      if(thread < 38211584) { /* Total Permutations, this frame: 1,048,576 ( 1 << 20 ) */
         seed[ 0] = Z_TIME[(thread & 15)];         // 2^4
         seed[ 1] = Z_AMB[(thread >> 4) & 15];     // 2^4
         seed[ 2] = 1;
         seed[ 3] = Z_ADJ[(thread >> 8) & 63];     // 2^6
         seed[ 4] = X_MASS[(thread >> 14) & 3];    // 2^2
         seed[ 5] = 1;
         seed[ 6] = Z_ING[(thread >> 16) & 15];    // 2^4
      } else
      if(thread < 40308736) { /* Total Permutations, this frame: 2,097,152 ( 1 << 21 ) */
         seed[ 0] = Z_TIME[(thread & 15)];         // 2^4
         seed[ 1] = Z_AMB[(thread >> 4) & 15];     // 2^4
         seed[ 2] = 1;
         seed[ 3] = Z_ADJ[(thread >> 8) & 63];     // 2^6
         seed[ 4] = Z_MASS[(thread >> 14) & 15];   // 2^4
         seed[ 5] = 1;
         seed[ 6] = Y_ING[(thread >> 18) & 7];     // 2^3
      } else
      if(thread < 41357312) { /* Total Permutations, this frame: 1,048,576 ( 1 << 20 ) */
         seed[ 0] = Z_TIME[(thread & 15)];         // 2^4
         seed[ 1] = Z_AMB[(thread >> 4) & 15];     // 2^4
         seed[ 2] = 1;
         seed[ 3] = Z_ADJ[(thread >> 8) & 63];     // 2^6
         seed[ 4] = Y_MASS[(thread >> 14) & 7];    // 2^3
         seed[ 5] = 1;
         seed[ 6] = Y_ING[(thread >> 17) & 7];     // 2^3
      } else
      if(thread < 41881600) { /* Total Permutations, this frame: 524,288 ( 1 << 19 ) */
         seed[ 0] = Z_TIME[(thread & 15)];         // 2^4
         seed[ 1] = Z_AMB[(thread >> 4) & 15];     // 2^4
         seed[ 2] = 1;
         seed[ 3] = Z_ADJ[(thread >> 8) & 63];     // 2^6
         seed[ 4] = X_MASS[(thread >> 14) & 3];    // 2^2
         seed[ 5] = 1;
         seed[ 6] = Y_ING[(thread >> 16) & 7];     // 2^3
      } else
      if(thread < 42405888) { /* Total Permutations, this frame: 524,288 ( 1 << 19  ) */
         seed[ 0] = Z_TIME[(thread & 15)];         // 2^4
         seed[ 1] = Z_AMB[(thread >> 4) & 15];     // 2^4
         seed[ 2] = 1;
         seed[ 3] = Z_ADJ[(thread >> 8) & 63];     // 2^6
         seed[ 4] = Z_MASS[(thread >> 14) & 15];   // 2^4
         seed[ 5] = 1;
         seed[ 6] = X_ING[(thread >> 18) & 1];     // 2^1
      } else
      if(thread < 42668032) { /* Total Permutations, this frame: 262,144 ( 1 << 18 ) */
         seed[ 0] = Z_TIME[(thread & 15)];         // 2^4
         seed[ 1] = Z_AMB[(thread >> 4) & 15];     // 2^4
         seed[ 2] = 1;
         seed[ 3] = Z_ADJ[(thread >> 8) & 63];     // 2^6
         seed[ 4] = Y_MASS[(thread >> 14) & 7];    // 2^3
         seed[ 5] = 1;
         seed[ 6] = X_ING[(thread >> 17) & 1];     // 2^1
      } else
      if(thread < 42799104) { /* Total Permutations, this frame: 131,072 ( 1 << 17 ) */
         seed[ 0] = Z_TIME[(thread & 15)];         // 2^4
         seed[ 1] = Z_AMB[(thread >> 4) & 15];     // 2^4
         seed[ 2] = 1;
         seed[ 3] = Z_ADJ[(thread >> 8) & 63];     // 2^6
         seed[ 4] = X_MASS[(thread >> 14) & 3];    // 2^2
         seed[ 5] = 1;
         seed[ 6] = X_ING[(thread >> 16) & 1];     // 2^1
      } else
      /* END Frame 7 */
      /* Frame 8 -> Split 2 ways */
      if(thread < 311234560) { /* Total Permutations, this frame: 268,435,456 ( 1 << 28 ) */
         seed[ 0] = Z_TIME[(thread & 15)];         // 2^4
         seed[ 1] = Z_AMB[(thread >> 4) & 15];     // 2^4
         seed[ 2] = 1;
         seed[ 3] = Z_PREP[(thread >> 8) & 3];     // 2^2
         seed[ 4] = 5;
         seed[ 5] = Z_ADJ[(thread >> 10) & 63];    // 2^6
         seed[ 6] = Z_NS[(thread >> 16) & 63];     // 2^6
         seed[ 7] = 3;
         seed[ 8] = 1;
         seed[ 9] = Z_INGADJ[(thread >> 22) & 63]; // 2^6
      } else
      if(thread < 445452288) { /* Total Permutations, this frame: 134,217,728 ( 1 << 27 ) */
         seed[ 0] = Z_TIME[(thread & 15)];         // 2^4
         seed[ 1] = Z_AMB[(thread >> 4) & 15];     // 2^4
         seed[ 2] = 1;
         seed[ 3] = Y_PREP[(thread >> 8) & 1];     // 2^1
         seed[ 4] = 5;
         seed[ 5] = Z_ADJ[(thread >> 9) & 63];     // 2^6
         seed[ 6] = Z_NS[(thread >> 15) & 63];     // 2^6
         seed[ 7] = 3;
         seed[ 8] = 1;
         seed[ 9] = Z_INGADJ[(thread >> 21) & 63]; // 2^6
      }

/* Below Two Frames are Valid, But Require 64-Bit Math: if extra entropy req'd.
   if( < thread <= ) { /* Total Permutations, this frame: 549,755,813,888
	seed[ 0] = Z_ING[(thread & 31)]; 
	seed[ 1] = Z_PREP[(thread << 5) & 7];
	seed[ 2] = Z_TIME[(thread << 8) & 15]; 
	seed[ 3] = Z_MASS[(thread << 12) & 31]; 
	seed[ 4] = 1;
        seed[ 5] = Z_MASS[(thread << 17) & 31]; 
	seed[ 6] = Z_ING[(thread << 22) & 31];  
	seed[ 7] = 3; 
	seed[ 8] = 1;
        seed[ 9] = 5; 
	seed[10] = Z_ADJ[(thread << 27) & 63];
	seed[11] = Z_NS[(thread << 33) & 63];
   }
   if( < thread <= ) { /* Total Permutations, this frame: 4,398,046,511,104
	seed[ 0] = Z_ING[(thread & 31)]; 
	seed[ 1] = Z_PREP[(thread << 5) & 7]; 
	seed[ 2] = 5; 
	seed[ 3] = Z_ADJ[(thread << 8) & 63]; 
	seed[ 4] = Z_NS[(thread << 14) & 63]; 
	seed[ 5] = 1;
        seed[ 6] = Z_MASS[(thread << 19) & 31]; 
	seed[ 7] = Z_ING[(thread << 24) & 31];  
	seed[ 8] = 3; 
	seed[ 9] = 1;
        seed[10] = 5; 
	seed[11] = Z_ADJ[(thread << 30) & 63]; 
	seed[12] = Z_NS[(thread << 36) & 63];
   }
End 64-bit Frames */
     
     /* store full nonce */
     #pragma unroll
     for (i = 0; i < 16; i++)
       nonce[i] = c_input32[i + 92];
     
     #pragma unroll
     for (i = 0; i < 16; i++)
       nonce[i+16] = seed[i];
     
       
     /*********************************************************/
     /* Hash 124 bytes of Block Trailer, including both seeds */
     /* Get the wizard to draw you a map to the princess!     */

     cuda_sha256_init(&ictx);

     /* update sha with the available block trailer data */
     cuda_sha256_update(&ictx, c_input32, 108);

     /* update sha with the second seed (16 bytes) */
     cuda_sha256_update(&ictx, seed, 16);

     /* finalise sha256 hash */
     cuda_sha256_final(&ictx, bt_hash);

      
     /*****************************************************/
     /* Determine the final tile based on selected nonce  */
     /* Time to find the princess!                        */
     
     /* determine first tile index */
     sm = bt_hash[0];
     for(i = 1; i < HASHLEN; i++)
       sm *= bt_hash[i];
     
     sm %= MAP;
       
     /* make <JUMP> tile jumps to find the final tile */
     for(j = 0; j < JUMP; j++) {
        sma[j] = sm;
        sm = cuda_next_index(sm, g_map, nonce);
     }


     /****************************************************************/
     /* Check the hash of the final tile produces the desired result */
     /* Search the castle for the princess!                          */
      
     cuda_sha256_init(&ictx);
     cuda_sha256_update(&ictx, bt_hash, HASHLEN);
     cuda_sha256_update(&ictx, &g_map[sm * TILE_LENGTH], TILE_LENGTH);
     cuda_sha256_final(&ictx, fhash);
     
     /* evaluate hash */
     for (x = i = j = n = 0; i < HASHLEN; i++) {
       x = fhash[i];
       if (x != 0) {
         for(j = 7; j > 0; j--) {
           x >>= 1;
           if(x == 0) {
             n += j;
             break;
           }
         }
         break;
       }
       n += 8;
     }
       
     if(n >= c_difficulty && !atomicExch(g_found, 1)) {
       /* PRINCESS FOUND! */
       #pragma unroll
       for (i = 0; i < 16; i++)
         g_seed[i] = seed[i];
     }
     
      /* Our princess is in another castle ! */
     
   }
}



extern "C" {

typedef struct __peach_cuda_ctx {
    byte curr_seed[16], next_seed[16];
    int *d_found, init;
    uint8_t *seed, *d_seed;
    uint8_t *input, *d_map;
    hipStream_t stream;
} PeachCudaCTX;

PeachCudaCTX ctx[63];    /* Max 63 GPUs Supported */
uint32_t threads = 1048576;
dim3 grid(4096);
dim3 block(256);
int *found;
byte *diff;
byte *phash;
byte gpuInit = 0;
byte bnum[8] = {0};
int nGPU = 0;

int init_cuda_peach(byte difficulty, byte *prevhash, byte *blocknumber) {
   /**
    * Definitions */
   int i;
   /**
    * Obtain and check system GPU count */
   hipGetDeviceCount(&nGPU);
   if(nGPU<1 || nGPU>63) return nGPU;
   /**
    * Allocate pinned host memory */
   hipHostMalloc(&diff, 1);
   hipHostMalloc(&found, 4);
   hipHostMalloc(&phash, 32);
   hipHostMalloc(&found, 4);
   /**
    * Copy immediate block data to pinned memory */
   memcpy(diff, &difficulty, 1);
   memset(found, 0, 4);
   memcpy(phash, prevhash, 32);
   /**
    * Initialize GPU context init variable incase
    * it holds a random number from memory */
   for (i = 0; i < nGPU && !gpuInit; i++)
      ctx[i].init = 0;
   gpuInit = 1;
   /**
    * Initialize GPU data asynchronously */
   for (i = 0; i < nGPU; i++) {
      hipSetDevice(i);
      /**
       * Create Stream */
      hipStreamCreate(&ctx[i].stream);
      /**
       * Allocate device memory */
      hipMalloc(&ctx[i].d_found, 4);
      hipMalloc(&ctx[i].d_seed, 16);
      /**
       * Allocate associated device-host memory */
      hipHostMalloc(&ctx[i].seed, 16);
      hipHostMalloc(&ctx[i].input, 108);
      /**
       * Copy immediate block data to device memory */
      hipMemcpyToSymbolAsync(HIP_SYMBOL(c_difficulty), diff, 1, 0,
                              hipMemcpyHostToDevice, ctx[i].stream);
      hipMemcpyToSymbolAsync(HIP_SYMBOL(c_phash), phash, 32, 0,
                              hipMemcpyHostToDevice, ctx[i].stream);
      /**
       * Set remaining device memory */
      hipMemsetAsync(ctx[i].d_found, 0, 4, ctx[i].stream);
      hipMemsetAsync(ctx[i].d_seed, 0, 16, ctx[i].stream);
      /**
       * Set initial round variables */
      ctx[i].next_seed[0] = 0;
      /**
       * If first init, setup map and cache */
      if(ctx[i].init == 0) {
         hipMalloc(&ctx[i].d_map, MAP_LENGTH);
         ctx[i].init = 1;
         /**
          * NOTE: The device MAP that holds the data of a map DOES NOT
          * explicitly get free()'d. The reason behind this is because
          * we reuse the map variable between blocks, and just rebuild
          * the map once every block. The GPU free's the MAP when the
          * program ends by default. This can be adjusted later. */
      }
      /**
       * (re)Build map if new block */
      if(memcmp(bnum, blocknumber, 8) != 0)
         cuda_build_map<<<4096, 256, 0, ctx[i].stream>>>
            (MAP,ctx[i].d_map);
   }
   /**
    * Check for any GPU initialization errors */
   for(i = 0; i < nGPU; i++) {
      hipSetDevice(i);
      hipStreamSynchronize(ctx[i].stream);
      cudaCheckError("init_cuda_peach()", i, __FILE__);
   }
   /**
    * Update block number */
   memcpy(bnum, blocknumber, 8);

   return nGPU;
}

void free_cuda_peach() {
   /**
    * Definitions */
   int i;
   /**
    * Free pinned host memory */
   hipHostFree(diff);
   hipHostFree(found);
   hipHostFree(phash);
   /**
    * Free GPU data */
   for (i = 0; i<nGPU; i++) {
      hipSetDevice(i);
      /**
       * Destroy Stream */
      hipStreamDestroy(ctx[i].stream);
      /**
       * Free device memory */
      hipFree(ctx[i].d_found);
      hipFree(ctx[i].d_seed);
      /**
       * Free associated device-host memory */
      hipHostFree(ctx[i].seed);
      hipHostFree(ctx[i].input);
      /**
       * Check for any GPU free() errors */
      cudaCheckError("free_cuda_peach()", i, __FILE__);
   }
}

extern byte *trigg_gen(byte *in);
extern char *trigg_expand2(byte *in, byte *out);

__host__ void cuda_peach(byte *bt, uint32_t *hps, byte *runflag)
{
   int i;
   uint64_t lastnHaiku, nHaiku = 0;
   time_t seconds = time(NULL);
   for( ; *runflag && *found == 0; ) {
      for (i=0; i<nGPU; i++) {
         /* Prepare next seed for GPU... */
         if(ctx[i].next_seed[0] == 0) {
            /* ... generate first GPU seed (and expand as Haiku) */
            trigg_gen(ctx[i].next_seed);

            /* ... and prepare round data */
            memcpy(ctx[i].input, bt, 92);
            memcpy(ctx[i].input+92, ctx[i].next_seed, 16);
         }
         /* Check if GPU has finished */
         hipSetDevice(i);
         if(hipStreamQuery(ctx[i].stream) == hipSuccess) {
            hipMemcpy(found, ctx[i].d_found, 4, hipMemcpyDeviceToHost);
            if(*found==1) { /* SOLVED A BLOCK! */
               hipMemcpy(ctx[i].seed, ctx[i].d_seed, 16, hipMemcpyDeviceToHost);
               memcpy(bt + 92, ctx[i].curr_seed, 16);
               memcpy(bt + 92 + 16, ctx[i].seed, 16);
               break;
            }
            /* Send new GPU round Data */
            hipMemcpyToSymbolAsync(HIP_SYMBOL(c_input32), ctx[i].input, 108, 0,
                                    hipMemcpyHostToDevice, ctx[i].stream);
            /* Start GPU round */
            cuda_find_peach<<<grid, block, 0, ctx[i].stream>>>(threads, MAP,
                           ctx[i].d_map, ctx[i].d_found, ctx[i].d_seed);

            /* Add to haiku count */
            nHaiku += threads;

            /* Store round vars aside for checks next loop */
            memcpy(ctx[i].curr_seed,ctx[i].next_seed,16);
            ctx[i].next_seed[0] = 0;
         }
         
         /* Waiting on GPU? ... */
         cudaCheckError("cuda_peach()", i, __FILE__);
      }
      
      /* Chill a bit if nothing is happening */
      if(lastnHaiku == nHaiku) usleep(100000);
      else lastnHaiku = nHaiku;
   }
    
   seconds = time(NULL) - seconds;
   if(seconds == 0) seconds = 1;
   nHaiku /= seconds;
   *hps = (uint32_t) nHaiku;
}


}
